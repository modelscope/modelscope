/*
 * SPDX-FileCopyrightText: Copyright (c) 2021-2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: LicenseRef-NvidiaProprietary
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
 */

#include "filtered_lrelu.cu"

// Template/kernel specializations for no signs mode (no gradients required).

// Full op, 32-bit indexing.
template filtered_lrelu_kernel_spec choose_filtered_lrelu_kernel<c10::Half, int32_t, false, false>(const filtered_lrelu_kernel_params& p, int sharedKB);
template filtered_lrelu_kernel_spec choose_filtered_lrelu_kernel<float,     int32_t, false, false>(const filtered_lrelu_kernel_params& p, int sharedKB);

// Full op, 64-bit indexing.
template filtered_lrelu_kernel_spec choose_filtered_lrelu_kernel<c10::Half, int64_t, false, false>(const filtered_lrelu_kernel_params& p, int sharedKB);
template filtered_lrelu_kernel_spec choose_filtered_lrelu_kernel<float,     int64_t, false, false>(const filtered_lrelu_kernel_params& p, int sharedKB);

// Activation/signs only for generic variant. 64-bit indexing.
template void* choose_filtered_lrelu_act_kernel<c10::Half, false, false>(void);
template void* choose_filtered_lrelu_act_kernel<float,     false, false>(void);
template void* choose_filtered_lrelu_act_kernel<double,    false, false>(void);

// Copy filters to constant memory.
template hipError_t copy_filters<false, false>(hipStream_t stream);
