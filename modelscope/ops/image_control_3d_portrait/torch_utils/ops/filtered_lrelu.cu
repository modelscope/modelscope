#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2021-2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: LicenseRef-NvidiaProprietary
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
 */

#include <c10/util/Half.h>
#include "filtered_lrelu.h"
#include <cstdint>

//------------------------------------------------------------------------
// Helpers.

enum // Filter modes.
{
    MODE_SUSD = 0,  // Separable upsampling, separable downsampling.
    MODE_FUSD = 1,  // Full upsampling, separable downsampling.
    MODE_SUFD = 2,  // Separable upsampling, full downsampling.
    MODE_FUFD = 3,  // Full upsampling, full downsampling.
};

template <class T> struct InternalType;
template <> struct InternalType<double>
{
    typedef double scalar_t; typedef double2 vec2_t; typedef double4 vec4_t;
    __device__ __forceinline__ static vec2_t zero_vec2(void) { return make_double2(0, 0); }
    __device__ __forceinline__ static vec4_t zero_vec4(void) { return make_double4(0, 0, 0, 0); }
    __device__ __forceinline__ static double clamp(double x, double c) { return fmin(fmax(x, -c), c); }
};
template <> struct InternalType<float>
{
    typedef float scalar_t; typedef float2 vec2_t; typedef float4 vec4_t;
    __device__ __forceinline__ static vec2_t zero_vec2(void) { return make_float2(0, 0); }
    __device__ __forceinline__ static vec4_t zero_vec4(void) { return make_float4(0, 0, 0, 0); }
    __device__ __forceinline__ static float clamp(float x, float c) { return fminf(fmaxf(x, -c), c); }
};
template <> struct InternalType<c10::Half>
{
    typedef float scalar_t; typedef float2 vec2_t; typedef float4 vec4_t;
    __device__ __forceinline__ static vec2_t zero_vec2(void) { return make_float2(0, 0); }
    __device__ __forceinline__ static vec4_t zero_vec4(void) { return make_float4(0, 0, 0, 0); }
    __device__ __forceinline__ static float clamp(float x, float c) { return fminf(fmaxf(x, -c), c); }
};

#define MIN(A, B)       ((A) < (B) ? (A) : (B))
#define MAX(A, B)       ((A) > (B) ? (A) : (B))
#define CEIL_DIV(A, B) (((B)==1) ? (A) : \
                        ((B)==2) ? ((int)((A)+1) >> 1) : \
                        ((B)==4) ? ((int)((A)+3) >> 2) : \
                        (((A) + ((A) > 0 ? (B) - 1 : 0)) / (B)))

// This works only up to blocks of size 256 x 256 and for all N that are powers of two.
template <int N> __device__ __forceinline__ void fast_div_mod(int& x, int& y, unsigned int i)
{
    if ((N & (N-1)) && N <= 256)
        y = (i * ((1<<24)/N + 1)) >> 24; // Assumes N <= 256, i < N*256.
    else
        y = i/N;

    x = i - y*N;
}

// Type cast stride before reading it.
template <class T> __device__ __forceinline__ T get_stride(const int64_t& x)
{
    return *reinterpret_cast<const T*>(&x);
}

//------------------------------------------------------------------------
// Filters, setup kernel, copying function.

#define MAX_FILTER_SIZE 32

// Combined up/down filter buffers so that transfer can be done with one copy.
__device__              float g_fbuf[2 * MAX_FILTER_SIZE * MAX_FILTER_SIZE]; // Filters in global memory, written by setup kernel.
__device__ __constant__ float c_fbuf[2 * MAX_FILTER_SIZE * MAX_FILTER_SIZE]; // Filters in constant memory, read by main kernel.

// Accessors to combined buffers to index up/down filters individually.
#define c_fu (c_fbuf)
#define c_fd (c_fbuf + MAX_FILTER_SIZE * MAX_FILTER_SIZE)
#define g_fu (g_fbuf)
#define g_fd (g_fbuf + MAX_FILTER_SIZE * MAX_FILTER_SIZE)

// Set up filters into global memory buffer.
static __global__ void setup_filters_kernel(filtered_lrelu_kernel_params p)
{
    for (int idx = threadIdx.x; idx < MAX_FILTER_SIZE * MAX_FILTER_SIZE; idx += blockDim.x)
    {
        int x, y;
        fast_div_mod<MAX_FILTER_SIZE>(x, y, idx);

        int fu_x = p.flip ? x : (p.fuShape.x - 1 - x);
        int fu_y = p.flip ? y : (p.fuShape.y - 1 - y);
        if (p.fuShape.y > 0)
            g_fu[idx] = (x >= p.fuShape.x || y >= p.fuShape.y) ? 0.0f : p.fu[fu_x * p.fuStride.x + fu_y * p.fuStride.y];
        else
            g_fu[idx] = (x >= p.fuShape.x || y > 0) ? 0.0f : p.fu[fu_x * p.fuStride.x];

        int fd_x = p.flip ? x : (p.fdShape.x - 1 - x);
        int fd_y = p.flip ? y : (p.fdShape.y - 1 - y);
        if (p.fdShape.y > 0)
            g_fd[idx] = (x >= p.fdShape.x || y >= p.fdShape.y) ? 0.0f : p.fd[fd_x * p.fdStride.x + fd_y * p.fdStride.y];
        else
            g_fd[idx] = (x >= p.fdShape.x || y > 0) ? 0.0f : p.fd[fd_x * p.fdStride.x];
    }
}

// Host function to copy filters written by setup kernel into constant buffer for main kernel.
template <bool, bool> static hipError_t copy_filters(hipStream_t stream)
{
    void* src = 0;
    hipError_t err = hipGetSymbolAddress(&src, HIP_SYMBOL(g_fbuf));
    if (err) return err;
    return hipMemcpyToSymbolAsync(HIP_SYMBOL(c_fbuf), src, 2 * MAX_FILTER_SIZE * MAX_FILTER_SIZE * sizeof(float), 0, hipMemcpyDeviceToDevice, stream);
}

//------------------------------------------------------------------------
// Coordinate spaces:
// - Relative to input tensor:      inX, inY, tileInX, tileInY
// - Relative to input tile:        relInX, relInY, tileInW, tileInH
// - Relative to upsampled tile:    relUpX, relUpY, tileUpW, tileUpH
// - Relative to output tile:       relOutX, relOutY, tileOutW, tileOutH
// - Relative to output tensor:     outX, outY, tileOutX, tileOutY
//
// Relationships between coordinate spaces:
// - inX = tileInX + relInX
// - inY = tileInY + relInY
// - relUpX = relInX * up + phaseInX
// - relUpY = relInY * up + phaseInY
// - relUpX = relOutX * down
// - relUpY = relOutY * down
// - outX = tileOutX + relOutX
// - outY = tileOutY + relOutY

extern __shared__ char s_buf_raw[]; // When sharedKB <= 48, allocate shared memory statically inside the kernel, otherwise use the externally allocated shared memory buffer.

template <class T, class index_t, int sharedKB, bool signWrite, bool signRead, int filterMode, int up, int fuSize, int down, int fdSize, int tileOutW, int tileOutH, int threadsPerBlock, bool enableXrep, bool enableWriteSkip>
static __global__ void filtered_lrelu_kernel(filtered_lrelu_kernel_params p)
{
    // Check that we don't try to support non-existing filter modes.
    static_assert(up   == 1 || up   == 2 || up   == 4, "only up=1, up=2, up=4 scales supported");
    static_assert(down == 1 || down == 2 || down == 4, "only down=1, down=2, down=4 scales supported");
    static_assert(fuSize >= up,   "upsampling filter size must be at least upsampling factor");
    static_assert(fdSize >= down, "downsampling filter size must be at least downsampling factor");
    static_assert(fuSize % up   == 0, "upsampling filter size must be divisible with upsampling factor");
    static_assert(fdSize % down == 0, "downsampling filter size must be divisible with downsampling factor");
    static_assert(fuSize <= MAX_FILTER_SIZE && fdSize <= MAX_FILTER_SIZE, "filter size greater than MAX_FILTER_SIZE");
    static_assert(up   != 1 || (fuSize == 1 && (filterMode == MODE_FUFD || filterMode == MODE_FUSD)), "up=1 supported only for 1x1 full filters");
    static_assert(down != 1 || (fdSize == 1 && (filterMode == MODE_FUFD || filterMode == MODE_SUFD)), "down=1 supported only for 1x1 full filters");
    static_assert(!(up   == 4 && (filterMode == MODE_FUFD || filterMode == MODE_FUSD)), "full filters not supported for up=4");
    static_assert(!(down == 4 && (filterMode == MODE_FUFD || filterMode == MODE_SUFD)), "full filters not supported for down=4");

    // Static definitions.
    typedef typename InternalType<T>::scalar_t scalar_t;
    typedef typename InternalType<T>::vec2_t vec2_t;
    typedef typename InternalType<T>::vec4_t vec4_t;
    const int tileUpW    = (tileOutW * down + (fdSize - 1) - (down - 1) + 3) & ~3;  // Upsampled tile width, rounded up to multiple of 4.
    const int tileUpH    = tileOutH * down + (fdSize - 1) - (down - 1);             // Upsampled tile height.
    const int tileInW    = CEIL_DIV(tileUpW  + (fuSize - 1), up);                   // Input tile width.
    const int tileInH    = CEIL_DIV(tileUpH  + (fuSize - 1), up);                   // Input tile height.
    const int tileUpH_up = CEIL_DIV(tileUpH, up) * up;                              // Upsampled tile height rounded up to a multiple of up.
    const int tileInH_up = CEIL_DIV(tileUpH_up + (fuSize - 1), up);                 // For allocations only, to avoid shared memory read overruns with up=2 and up=4.

    // Merge 1x1 downsampling into last upsampling step for upf1 and ups2.
    const bool downInline = (down == 1) && ((up == 1 && filterMode == MODE_FUFD) || (up == 2 && filterMode == MODE_SUFD));

    // Sizes of logical buffers.
    const int szIn    = tileInH_up * tileInW;
    const int szUpX   = tileInH_up * tileUpW;
    const int szUpXY  = downInline ? 0 : (tileUpH * tileUpW);
    const int szDownX = tileUpH * tileOutW;

    // Sizes for shared memory arrays.
    const int s_buf0_size_base =
        (filterMode == MODE_SUSD) ? MAX(szIn, szUpXY) :
        (filterMode == MODE_FUSD) ? MAX(szIn, szDownX) :
        (filterMode == MODE_SUFD) ? MAX(szIn, szUpXY) :
        (filterMode == MODE_FUFD) ? szIn :
        -1;
    const int s_buf1_size_base =
        (filterMode == MODE_SUSD) ? MAX(szUpX, szDownX) :
        (filterMode == MODE_FUSD) ? szUpXY :
        (filterMode == MODE_SUFD) ? szUpX  :
        (filterMode == MODE_FUFD) ? szUpXY :
        -1;

    // Ensure U128 alignment.
    const int s_buf0_size = (s_buf0_size_base + 3) & ~3;
    const int s_buf1_size = (s_buf1_size_base + 3) & ~3;

    // Check at compile time that we don't use too much shared memory.
    static_assert((s_buf0_size + s_buf1_size) * sizeof(scalar_t) <= (sharedKB << 10), "shared memory overflow");

    // Declare shared memory arrays.
    scalar_t* s_buf0;
    scalar_t* s_buf1;
    if (sharedKB <= 48)
    {
        // Allocate shared memory arrays here.
        __shared__ scalar_t s_buf0_st[(sharedKB > 48) ? (1<<24) : (s_buf0_size + s_buf1_size)]; // Prevent launching if this isn't optimized away when unused.
        s_buf0 = s_buf0_st;
        s_buf1 = s_buf0 + s_buf0_size;
    }
    else
    {
        // Use the dynamically allocated shared memory array.
        s_buf0 = (scalar_t*)s_buf_raw;
        s_buf1 = s_buf0 + s_buf0_size;
    }

    // Pointers to the buffers.
    scalar_t* s_tileIn;       // Input tile:                      [relInX * tileInH + relInY]
    scalar_t* s_tileUpX;      // After horizontal upsampling:     [relInY * tileUpW + relUpX]
    scalar_t* s_tileUpXY;     // After upsampling:                [relUpY * tileUpW + relUpX]
    scalar_t* s_tileDownX;    // After horizontal downsampling:   [relUpY * tileOutW + relOutX]
    if (filterMode == MODE_SUSD)
    {
        s_tileIn    = s_buf0;
        s_tileUpX   = s_buf1;
        s_tileUpXY  = s_buf0;
        s_tileDownX = s_buf1;
    }
    else if (filterMode == MODE_FUSD)
    {
        s_tileIn    = s_buf0;
        s_tileUpXY  = s_buf1;
        s_tileDownX = s_buf0;
    }
    else if (filterMode == MODE_SUFD)
    {
        s_tileIn    = s_buf0;
        s_tileUpX   = s_buf1;
        s_tileUpXY  = s_buf0;
    }
    else if (filterMode == MODE_FUFD)
    {
        s_tileIn    = s_buf0;
        s_tileUpXY  = s_buf1;
    }

    // Allow large grids in z direction via per-launch offset.
    int channelIdx = blockIdx.z + p.blockZofs;
    int batchIdx = channelIdx / p.yShape.z;
    channelIdx -= batchIdx * p.yShape.z;

    // Offset to output feature map. In bytes.
    index_t mapOfsOut = channelIdx * get_stride<index_t>(p.yStride.z) + batchIdx * get_stride<index_t>(p.yStride.w);

    // Sign shift amount.
    uint32_t signXo = ((threadIdx.x + p.sOfs.x) << 1) & 6;

    // Inner tile loop.
    #pragma unroll 1
    for (int tileIdx = 0; !enableXrep || (tileIdx < MIN(p.tilesXrep, p.tilesXdim - p.tilesXrep * blockIdx.y)); tileIdx++)
    {
        // Locate output tile.
        int tileX = enableXrep ? blockIdx.y * p.tilesXrep + tileIdx : blockIdx.x;
        int tileOutX = tileX * tileOutW;
        int tileOutY = (enableXrep ? blockIdx.x : blockIdx.y) * tileOutH;

        // Locate input tile.
        int tmpX = tileOutX * down - p.pad0.x;
        int tmpY = tileOutY * down - p.pad0.y;
        int tileInX = CEIL_DIV(tmpX, up);
        int tileInY = CEIL_DIV(tmpY, up);
        const int phaseInX = tileInX * up - tmpX;
        const int phaseInY = tileInY * up - tmpY;

        // Extra sync if input and output buffers are the same and we are not on first tile.
        if (enableXrep && tileIdx > 0 && (filterMode == MODE_FUSD || (filterMode == MODE_SUFD && !downInline) || (filterMode == MODE_FUFD && downInline)))
            __syncthreads();

        // Load input tile & apply bias. Unrolled.
        scalar_t b = (scalar_t)*(const T*)((const char*)p.b + (channelIdx * get_stride<index_t>(p.bStride)));
        index_t mapOfsIn = channelIdx * get_stride<index_t>(p.xStride.z) + batchIdx * get_stride<index_t>(p.xStride.w);
        int idx = threadIdx.x;
        const int loopCountIN = CEIL_DIV(tileInW * tileInH, threadsPerBlock);
        #pragma unroll
        for (int loop = 0; loop < loopCountIN; loop++)
        {
            int relInX, relInY;
            fast_div_mod<tileInW>(relInX, relInY, idx);
            int inX = tileInX + relInX;
            int inY = tileInY + relInY;
            scalar_t v = 0;

            if ((uint32_t)inX < p.xShape.x && (uint32_t)inY < p.xShape.y)
                v = (scalar_t)*((const T*)((const char*)p.x + (inX * get_stride<index_t>(p.xStride.x) + inY * get_stride<index_t>(p.xStride.y) + mapOfsIn))) + b;

            bool skip = (loop == loopCountIN-1) && (idx >= tileInW * tileInH);
            if (!skip)
                s_tileIn[idx] = v;

            idx += threadsPerBlock;
        }

        if (filterMode == MODE_SUSD || filterMode == MODE_SUFD) // Separable upsampling filter.
        {
            // Horizontal upsampling.
            __syncthreads();
            if (up == 4)
            {
                for (int idx = threadIdx.x*up; idx < tileUpW * tileInH; idx += blockDim.x*up)
                {
                    int relUpX0, relInY;
                    fast_div_mod<tileUpW>(relUpX0, relInY, idx);
                    int relInX0 = relUpX0 / up;
                    int src0 = relInX0 + tileInW * relInY;
                    int dst = relInY * tileUpW + relUpX0;
                    vec4_t v = InternalType<T>::zero_vec4();
                    scalar_t a = s_tileIn[src0];
                    if (phaseInX == 0)
                    {
                        #pragma unroll
                        for (int step = 0; step < fuSize / up; step++)
                        {
                            v.x += a * (scalar_t)c_fu[step * up + 0];
                            a = s_tileIn[src0 + step + 1];
                            v.y += a * (scalar_t)c_fu[step * up + 3];
                            v.z += a * (scalar_t)c_fu[step * up + 2];
                            v.w += a * (scalar_t)c_fu[step * up + 1];
                        }
                    }
                    else if (phaseInX == 1)
                    {
                        #pragma unroll
                        for (int step = 0; step < fuSize / up; step++)
                        {
                            v.x += a * (scalar_t)c_fu[step * up + 1];
                            v.y += a * (scalar_t)c_fu[step * up + 0];
                            a = s_tileIn[src0 + step + 1];
                            v.z += a * (scalar_t)c_fu[step * up + 3];
                            v.w += a * (scalar_t)c_fu[step * up + 2];
                        }
                    }
                    else if (phaseInX == 2)
                    {
                        #pragma unroll
                        for (int step = 0; step < fuSize / up; step++)
                        {
                            v.x += a * (scalar_t)c_fu[step * up + 2];
                            v.y += a * (scalar_t)c_fu[step * up + 1];
                            v.z += a * (scalar_t)c_fu[step * up + 0];
                            a = s_tileIn[src0 + step + 1];
                            v.w += a * (scalar_t)c_fu[step * up + 3];
                        }
                    }
                    else // (phaseInX == 3)
                    {
                        #pragma unroll
                        for (int step = 0; step < fuSize / up; step++)
                        {
                            v.x += a * (scalar_t)c_fu[step * up + 3];
                            v.y += a * (scalar_t)c_fu[step * up + 2];
                            v.z += a * (scalar_t)c_fu[step * up + 1];
                            v.w += a * (scalar_t)c_fu[step * up + 0];
                            a = s_tileIn[src0 + step + 1];
                        }
                    }
                    s_tileUpX[dst+0] = v.x;
                    s_tileUpX[dst+1] = v.y;
                    s_tileUpX[dst+2] = v.z;
                    s_tileUpX[dst+3] = v.w;
                }
            }
            else if (up == 2)
            {
                bool p0 = (phaseInX == 0);
                for (int idx = threadIdx.x*up; idx < tileUpW * tileInH; idx += blockDim.x*up)
                {
                    int relUpX0, relInY;
                    fast_div_mod<tileUpW>(relUpX0, relInY, idx);
                    int relInX0 = relUpX0 / up;
                    int src0 = relInX0 + tileInW * relInY;
                    int dst = relInY * tileUpW + relUpX0;
                    vec2_t v = InternalType<T>::zero_vec2();
                    scalar_t a = s_tileIn[src0];
                    if (p0) // (phaseInX == 0)
                    {
                        #pragma unroll
                        for (int step = 0; step < fuSize / up; step++)
                        {
                            v.x += a * (scalar_t)c_fu[step * up + 0];
                            a = s_tileIn[src0 + step + 1];
                            v.y += a * (scalar_t)c_fu[step * up + 1];
                        }
                    }
                    else // (phaseInX == 1)
                    {
                        #pragma unroll
                        for (int step = 0; step < fuSize / up; step++)
                        {
                            v.x += a * (scalar_t)c_fu[step * up + 1];
                            v.y += a * (scalar_t)c_fu[step * up + 0];
                            a = s_tileIn[src0 + step + 1];
                        }
                    }
                    s_tileUpX[dst+0] = v.x;
                    s_tileUpX[dst+1] = v.y;
                }
            }

            // Vertical upsampling & nonlinearity.

            __syncthreads();
            int groupMask = 15 << ((threadIdx.x & 31) & ~3);
            int minY = tileOutY ? (tileOutY - tileOutH) * down + tileUpH : 0; // Skip already written signs.
            int sShapeMaxY = MIN(p.sShape.y, tileOutY * down + tileUpH); // Avoid out-of-tile sign writes.
            if (up == 4)
            {
                minY -= 3; // Adjust according to block height.
                for (int idx = threadIdx.x; idx < tileUpW * tileUpH_up / up; idx += blockDim.x)
                {
                    int relUpX, relInY0;
                    fast_div_mod<tileUpW>(relUpX, relInY0, idx);
                    int relUpY0 = relInY0 * up;
                    int src0 = relInY0 * tileUpW + relUpX;
                    int dst = relUpY0 * tileUpW + relUpX;
                    vec4_t v = InternalType<T>::zero_vec4();

                    scalar_t a = s_tileUpX[src0];
                    if (phaseInY == 0)
                    {
                        #pragma unroll
                        for (int step = 0; step < fuSize / up; step++)
                        {
                            v.x += a * (scalar_t)c_fu[step * up + 0];
                            a = s_tileUpX[src0 + (step + 1) * tileUpW];
                            v.y += a * (scalar_t)c_fu[step * up + 3];
                            v.z += a * (scalar_t)c_fu[step * up + 2];
                            v.w += a * (scalar_t)c_fu[step * up + 1];
                        }
                    }
                    else if (phaseInY == 1)
                    {
                        #pragma unroll
                        for (int step = 0; step < fuSize / up; step++)
                        {
                            v.x += a * (scalar_t)c_fu[step * up + 1];
                            v.y += a * (scalar_t)c_fu[step * up + 0];
                            a = s_tileUpX[src0 + (step + 1) * tileUpW];
                            v.z += a * (scalar_t)c_fu[step * up + 3];
                            v.w += a * (scalar_t)c_fu[step * up + 2];
                        }
                    }
                    else if (phaseInY == 2)
                    {
                        #pragma unroll
                        for (int step = 0; step < fuSize / up; step++)
                        {
                            v.x += a * (scalar_t)c_fu[step * up + 2];
                            v.y += a * (scalar_t)c_fu[step * up + 1];
                            v.z += a * (scalar_t)c_fu[step * up + 0];
                            a = s_tileUpX[src0 + (step + 1) * tileUpW];
                            v.w += a * (scalar_t)c_fu[step * up + 3];
                        }
                    }
                    else // (phaseInY == 3)
                    {
                        #pragma unroll
                        for (int step = 0; step < fuSize / up; step++)
                        {
                            v.x += a * (scalar_t)c_fu[step * up + 3];
                            v.y += a * (scalar_t)c_fu[step * up + 2];
                            v.z += a * (scalar_t)c_fu[step * up + 1];
                            v.w += a * (scalar_t)c_fu[step * up + 0];
                            a = s_tileUpX[src0 + (step + 1) * tileUpW];
                        }
                    }

                    int x = tileOutX * down + relUpX;
                    int y = tileOutY * down + relUpY0;
                    int signX = x + p.sOfs.x;
                    int signY = y + p.sOfs.y;
                    int signZ = blockIdx.z + p.blockZofs;
                    int signXb = signX >> 2;
                    index_t si0 = signXb + p.sShape.x * (signY + (index_t)p.sShape.y * signZ);
                    index_t si1 = si0 + p.sShape.x;
                    index_t si2 = si0 + p.sShape.x * 2;
                    index_t si3 = si0 + p.sShape.x * 3;

                    v.x *= (scalar_t)((float)up * (float)up * p.gain);
                    v.y *= (scalar_t)((float)up * (float)up * p.gain);
                    v.z *= (scalar_t)((float)up * (float)up * p.gain);
                    v.w *= (scalar_t)((float)up * (float)up * p.gain);

                    if (signWrite)
                    {
                        if (!enableWriteSkip)
                        {
                            // Determine and write signs.
                            int sx = __float_as_uint(v.x) >> 31 <<  0;
                            int sy = __float_as_uint(v.y) >> 31 <<  8;
                            int sz = __float_as_uint(v.z) >> 31 << 16;
                            int sw = __float_as_uint(v.w) >> 31 << 24;
                            if (sx) v.x *= p.slope;
                            if (sy) v.y *= p.slope;
                            if (sz) v.z *= p.slope;
                            if (sw) v.w *= p.slope;
                            if (fabsf(v.x) > p.clamp) { sx = 2 <<  0; v.x = InternalType<T>::clamp(v.x, p.clamp); }
                            if (fabsf(v.y) > p.clamp) { sy = 2 <<  8; v.y = InternalType<T>::clamp(v.y, p.clamp); }
                            if (fabsf(v.z) > p.clamp) { sz = 2 << 16; v.z = InternalType<T>::clamp(v.z, p.clamp); }
                            if (fabsf(v.w) > p.clamp) { sw = 2 << 24; v.w = InternalType<T>::clamp(v.w, p.clamp); }

                            if ((uint32_t)signXb < p.swLimit && signY >= minY)
                            {
                                // Combine signs.
                                uint32_t s = sx + sy + sw + sz;
                                s <<= (signX & 3) << 1;
                                s |= __shfl_xor_sync(groupMask, s, 1);
                                s |= __shfl_xor_sync(groupMask, s, 2);

                                // Write signs.
                                if ((uint32_t)(signY + 0) < sShapeMaxY) { p.s[si0] = (unsigned char)(s >>  0); }
                                if ((uint32_t)(signY + 1) < sShapeMaxY) { p.s[si1] = (unsigned char)(s >>  8); }
                                if ((uint32_t)(signY + 2) < sShapeMaxY) { p.s[si2] = (unsigned char)(s >> 16); }
                                if ((uint32_t)(signY + 3) < sShapeMaxY) { p.s[si3] = (unsigned char)(s >> 24); }
                            }
                        }
                        else
                        {
                            // Determine and write signs.
                            if ((uint32_t)signXb < p.swLimit && signY >= minY)
                            {
                                int sx = __float_as_uint(v.x) >> 31 <<  0;
                                int sy = __float_as_uint(v.y) >> 31 <<  8;
                                int sz = __float_as_uint(v.z) >> 31 << 16;
                                int sw = __float_as_uint(v.w) >> 31 << 24;
                                if (sx) v.x *= p.slope;
                                if (sy) v.y *= p.slope;
                                if (sz) v.z *= p.slope;
                                if (sw) v.w *= p.slope;
                                if (fabsf(v.x) > p.clamp) { sx = 2 <<  0; v.x = InternalType<T>::clamp(v.x, p.clamp); }
                                if (fabsf(v.y) > p.clamp) { sy = 2 <<  8; v.y = InternalType<T>::clamp(v.y, p.clamp); }
                                if (fabsf(v.z) > p.clamp) { sz = 2 << 16; v.z = InternalType<T>::clamp(v.z, p.clamp); }
                                if (fabsf(v.w) > p.clamp) { sw = 2 << 24; v.w = InternalType<T>::clamp(v.w, p.clamp); }

                                // Combine signs.
                                uint32_t s = sx + sy + sw + sz;
                                s <<= (signX & 3) << 1;
                                s |= __shfl_xor_sync(groupMask, s, 1);
                                s |= __shfl_xor_sync(groupMask, s, 2);

                                // Write signs.
                                if ((uint32_t)(signY + 0) < sShapeMaxY) { p.s[si0] = (unsigned char)(s >>  0); }
                                if ((uint32_t)(signY + 1) < sShapeMaxY) { p.s[si1] = (unsigned char)(s >>  8); }
                                if ((uint32_t)(signY + 2) < sShapeMaxY) { p.s[si2] = (unsigned char)(s >> 16); }
                                if ((uint32_t)(signY + 3) < sShapeMaxY) { p.s[si3] = (unsigned char)(s >> 24); }
                            }
                            else
                            {
                                // Just compute the values.
                                if (v.x < 0.f) v.x *= p.slope; v.x = InternalType<T>::clamp(v.x, p.clamp);
                                if (v.y < 0.f) v.y *= p.slope; v.y = InternalType<T>::clamp(v.y, p.clamp);
                                if (v.z < 0.f) v.z *= p.slope; v.z = InternalType<T>::clamp(v.z, p.clamp);
                                if (v.w < 0.f) v.w *= p.slope; v.w = InternalType<T>::clamp(v.w, p.clamp);
                            }
                        }
                    }
                    else if (signRead) // Read signs and apply.
                    {
                        if ((uint32_t)signXb < p.swLimit)
                        {
                            int ss = (signX & 3) << 1;
                            if ((uint32_t)(signY + 0) < p.sShape.y) { int s = p.s[si0] >> ss; if (s & 1) v.x *= p.slope; if (s & 2) v.x = 0.f; }
                            if ((uint32_t)(signY + 1) < p.sShape.y) { int s = p.s[si1] >> ss; if (s & 1) v.y *= p.slope; if (s & 2) v.y = 0.f; }
                            if ((uint32_t)(signY + 2) < p.sShape.y) { int s = p.s[si2] >> ss; if (s & 1) v.z *= p.slope; if (s & 2) v.z = 0.f; }
                            if ((uint32_t)(signY + 3) < p.sShape.y) { int s = p.s[si3] >> ss; if (s & 1) v.w *= p.slope; if (s & 2) v.w = 0.f; }
                        }
                    }
                    else // Forward pass with no sign write.
                    {
                        if (v.x < 0.f) v.x *= p.slope; v.x = InternalType<T>::clamp(v.x, p.clamp);
                        if (v.y < 0.f) v.y *= p.slope; v.y = InternalType<T>::clamp(v.y, p.clamp);
                        if (v.z < 0.f) v.z *= p.slope; v.z = InternalType<T>::clamp(v.z, p.clamp);
                        if (v.w < 0.f) v.w *= p.slope; v.w = InternalType<T>::clamp(v.w, p.clamp);
                    }

                    s_tileUpXY[dst + 0 * tileUpW] = v.x;
                    if (relUpY0 + 1 < tileUpH) s_tileUpXY[dst + 1 * tileUpW] = v.y;
                    if (relUpY0 + 2 < tileUpH) s_tileUpXY[dst + 2 * tileUpW] = v.z;
                    if (relUpY0 + 3 < tileUpH) s_tileUpXY[dst + 3 * tileUpW] = v.w;
                }
            }
            else if (up == 2)
            {
                minY -= 1; // Adjust according to block height.
                for (int idx = threadIdx.x; idx < tileUpW * tileUpH_up / up; idx += blockDim.x)
                {
                    int relUpX, relInY0;
                    fast_div_mod<tileUpW>(relUpX, relInY0, idx);
                    int relUpY0 = relInY0 * up;
                    int src0 = relInY0 * tileUpW + relUpX;
                    int dst = relUpY0 * tileUpW + relUpX;
                    vec2_t v = InternalType<T>::zero_vec2();

                    scalar_t a = s_tileUpX[src0];
                    if (phaseInY == 0)
                    {
                        #pragma unroll
                        for (int step = 0; step < fuSize / up; step++)
                        {
                            v.x += a * (scalar_t)c_fu[step * up + 0];
                            a = s_tileUpX[src0 + (step + 1) * tileUpW];
                            v.y += a * (scalar_t)c_fu[step * up + 1];
                        }
                    }
                    else // (phaseInY == 1)
                    {
                        #pragma unroll
                        for (int step = 0; step < fuSize / up; step++)
                        {
                            v.x += a * (scalar_t)c_fu[step * up + 1];
                            v.y += a * (scalar_t)c_fu[step * up + 0];
                            a = s_tileUpX[src0 + (step + 1) * tileUpW];
                        }
                    }

                    int x = tileOutX * down + relUpX;
                    int y = tileOutY * down + relUpY0;
                    int signX = x + p.sOfs.x;
                    int signY = y + p.sOfs.y;
                    int signZ = blockIdx.z + p.blockZofs;
                    int signXb = signX >> 2;
                    index_t si0 = signXb + p.sShape.x * (signY + (index_t)p.sShape.y * signZ);
                    index_t si1 = si0 + p.sShape.x;

                    v.x *= (scalar_t)((float)up * (float)up * p.gain);
                    v.y *= (scalar_t)((float)up * (float)up * p.gain);

                    if (signWrite)
                    {
                        if (!enableWriteSkip)
                        {
                            // Determine and write signs.
                            int sx = __float_as_uint(v.x) >> 31 << 0;
                            int sy = __float_as_uint(v.y) >> 31 << 8;
                            if (sx) v.x *= p.slope;
                            if (sy) v.y *= p.slope;
                            if (fabsf(v.x) > p.clamp) { sx = 2 << 0; v.x = InternalType<T>::clamp(v.x, p.clamp); }
                            if (fabsf(v.y) > p.clamp) { sy = 2 << 8; v.y = InternalType<T>::clamp(v.y, p.clamp); }

                            if ((uint32_t)signXb < p.swLimit && signY >= minY)
                            {
                                // Combine signs.
                                int s = sx + sy;
                                s <<= signXo;
                                s |= __shfl_xor_sync(groupMask, s, 1);
                                s |= __shfl_xor_sync(groupMask, s, 2);

                                // Write signs.
                                if ((uint32_t)(signY + 0) < sShapeMaxY) { p.s[si0] = (unsigned char)(s >>  0); }
                                if ((uint32_t)(signY + 1) < sShapeMaxY) { p.s[si1] = (unsigned char)(s >>  8); }
                            }
                        }
                        else
                        {
                            // Determine and write signs.
                            if ((uint32_t)signXb < p.swLimit && signY >= minY)
                            {
                                int sx = __float_as_uint(v.x) >> 31 << 0;
                                int sy = __float_as_uint(v.y) >> 31 << 8;
                                if (sx) v.x *= p.slope;
                                if (sy) v.y *= p.slope;
                                if (fabsf(v.x) > p.clamp) { sx = 2 << 0; v.x = InternalType<T>::clamp(v.x, p.clamp); }
                                if (fabsf(v.y) > p.clamp) { sy = 2 << 8; v.y = InternalType<T>::clamp(v.y, p.clamp); }

                                // Combine signs.
                                int s = sx + sy;
                                s <<= signXo;
                                s |= __shfl_xor_sync(groupMask, s, 1);
                                s |= __shfl_xor_sync(groupMask, s, 2);

                                // Write signs.
                                if ((uint32_t)(signY + 0) < sShapeMaxY) { p.s[si0] = (unsigned char)(s >>  0); }
                                if ((uint32_t)(signY + 1) < sShapeMaxY) { p.s[si1] = (unsigned char)(s >>  8); }
                            }
                            else
                            {
                                // Just compute the values.
                                if (v.x < 0.f) v.x *= p.slope; v.x = InternalType<T>::clamp(v.x, p.clamp);
                                if (v.y < 0.f) v.y *= p.slope; v.y = InternalType<T>::clamp(v.y, p.clamp);
                            }
                        }
                    }
                    else if (signRead) // Read signs and apply.
                    {
                        if ((uint32_t)signXb < p.swLimit)
                        {
                            if ((uint32_t)(signY + 0) < p.sShape.y) { int s = p.s[si0] >> signXo; if (s & 1) v.x *= p.slope; if (s & 2) v.x = 0.f; }
                            if ((uint32_t)(signY + 1) < p.sShape.y) { int s = p.s[si1] >> signXo; if (s & 1) v.y *= p.slope; if (s & 2) v.y = 0.f; }
                        }
                    }
                    else // Forward pass with no sign write.
                    {
                        if (v.x < 0.f) v.x *= p.slope; v.x = InternalType<T>::clamp(v.x, p.clamp);
                        if (v.y < 0.f) v.y *= p.slope; v.y = InternalType<T>::clamp(v.y, p.clamp);
                    }

                    if (!downInline)
                    {
                        // Write into temporary buffer.
                        s_tileUpXY[dst] = v.x;
                        if (relUpY0 < tileUpH - 1)
                            s_tileUpXY[dst + tileUpW] = v.y;
                    }
                    else
                    {
                        // Write directly into output buffer.
                        if ((uint32_t)x < p.yShape.x)
                        {
                            int ymax = MIN(p.yShape.y, tileUpH + tileOutY * down);
                            index_t ofs = x * get_stride<index_t>(p.yStride.x) + y * get_stride<index_t>(p.yStride.y) + mapOfsOut;
                            if ((uint32_t)y + 0 < p.yShape.y) *((T*)((char*)p.y + ofs)) = (T)(v.x * (scalar_t)c_fd[0]);
                            if ((uint32_t)y + 1 < ymax) *((T*)((char*)p.y + ofs + get_stride<index_t>(p.yStride.y))) = (T)(v.y * (scalar_t)c_fd[0]);
                        }
                    }
                }
            }
        }
        else if (filterMode == MODE_FUSD || filterMode == MODE_FUFD)
        {
            // Full upsampling filter.

            if (up == 2)
            {
                // 2 x 2-wide.
                __syncthreads();
                int minY = tileOutY ? (tileOutY - tileOutH) * down + tileUpH + p.sOfs.y : 0; // Skip already written signs.
                for (int idx = threadIdx.x * 4; idx < tileUpW * tileUpH; idx += blockDim.x * 4)
                {
                    int relUpX0, relUpY0;
                    fast_div_mod<tileUpW>(relUpX0, relUpY0, idx);
                    int relInX0 = CEIL_DIV(relUpX0 - phaseInX, up);
                    int relInY0 = CEIL_DIV(relUpY0 - phaseInY, up);
                    int src0 = relInX0 + tileInW * relInY0;
                    int tap0y = (relInY0 * up + phaseInY - relUpY0);

                    #define X_LOOP(TAPY, PX) \
                        for (int sx = 0; sx < fuSize / up; sx++) \
                        { \
                            v.x += a * (scalar_t)c_fu[(sx * up + (((PX) - 0) & (up - 1))) + (sy * up + (TAPY)) * MAX_FILTER_SIZE]; \
                            v.z += b * (scalar_t)c_fu[(sx * up + (((PX) - 0) & (up - 1))) + (sy * up + (TAPY)) * MAX_FILTER_SIZE]; if ((PX) == 0) { a = b; b = s_tileIn[src0 + 2 + sx + sy * tileInW]; } \
                            v.y += a * (scalar_t)c_fu[(sx * up + (((PX) - 1) & (up - 1))) + (sy * up + (TAPY)) * MAX_FILTER_SIZE]; \
                            v.w += b * (scalar_t)c_fu[(sx * up + (((PX) - 1) & (up - 1))) + (sy * up + (TAPY)) * MAX_FILTER_SIZE]; if ((PX) == 1) { a = b; b = s_tileIn[src0 + 2 + sx + sy * tileInW]; } \
                        }

                    vec4_t v = InternalType<T>::zero_vec4();
                    if (tap0y == 0 && phaseInX == 0)
                        #pragma unroll
                        for (int sy = 0; sy < fuSize / up; sy++) { scalar_t a = s_tileIn[src0 + sy * tileInW]; scalar_t b = s_tileIn[src0 + sy * tileInW + 1];
                            #pragma unroll
                            X_LOOP(0, 0) }
                    if (tap0y == 0 && phaseInX == 1)
                        #pragma unroll
                        for (int sy = 0; sy < fuSize / up; sy++) { scalar_t a = s_tileIn[src0 + sy * tileInW]; scalar_t b = s_tileIn[src0 + sy * tileInW + 1];
                            #pragma unroll
                            X_LOOP(0, 1) }
                    if (tap0y == 1 && phaseInX == 0)
                        #pragma unroll
                        for (int sy = 0; sy < fuSize / up; sy++) { scalar_t a = s_tileIn[src0 + sy * tileInW]; scalar_t b = s_tileIn[src0 + sy * tileInW + 1];
                            #pragma unroll
                            X_LOOP(1, 0) }
                    if (tap0y == 1 && phaseInX == 1)
                        #pragma unroll
                        for (int sy = 0; sy < fuSize / up; sy++) { scalar_t a = s_tileIn[src0 + sy * tileInW]; scalar_t b = s_tileIn[src0 + sy * tileInW + 1];
                            #pragma unroll
                            X_LOOP(1, 1) }

                    #undef X_LOOP

                    int x = tileOutX * down + relUpX0;
                    int y = tileOutY * down + relUpY0;
                    int signX = x + p.sOfs.x;
                    int signY = y + p.sOfs.y;
                    int signZ = blockIdx.z + p.blockZofs;
                    int signXb = signX >> 2;
                    index_t si = signXb + p.sShape.x * (signY + (index_t)p.sShape.y * signZ);

                    v.x *= (scalar_t)((float)up * (float)up * p.gain);
                    v.y *= (scalar_t)((float)up * (float)up * p.gain);
                    v.z *= (scalar_t)((float)up * (float)up * p.gain);
                    v.w *= (scalar_t)((float)up * (float)up * p.gain);

                    if (signWrite)
                    {
                        if (!enableWriteSkip)
                        {
                            // Determine and write signs.
                            int sx = __float_as_uint(v.x) >> 31;
                            int sy = __float_as_uint(v.y) >> 31;
                            int sz = __float_as_uint(v.z) >> 31;
                            int sw = __float_as_uint(v.w) >> 31;
                            if (sx) v.x *= p.slope; if (fabsf(v.x) > p.clamp) { sx = 2; v.x = InternalType<T>::clamp(v.x, p.clamp); }
                            if (sy) v.y *= p.slope; if (fabsf(v.y) > p.clamp) { sy = 2; v.y = InternalType<T>::clamp(v.y, p.clamp); }
                            if (sz) v.z *= p.slope; if (fabsf(v.z) > p.clamp) { sz = 2; v.z = InternalType<T>::clamp(v.z, p.clamp); }
                            if (sw) v.w *= p.slope; if (fabsf(v.w) > p.clamp) { sw = 2; v.w = InternalType<T>::clamp(v.w, p.clamp); }

                            if ((uint32_t)signXb < p.swLimit && (uint32_t)signY < p.sShape.y && signY >= minY)
                            {
                                p.s[si] = sx + (sy << 2) + (sz << 4) + (sw << 6);
                            }
                        }
                        else
                        {
                            // Determine and write signs.
                            if ((uint32_t)signXb < p.swLimit && (uint32_t)signY < p.sShape.y && signY >= minY)
                            {
                                int sx = __float_as_uint(v.x) >> 31;
                                int sy = __float_as_uint(v.y) >> 31;
                                int sz = __float_as_uint(v.z) >> 31;
                                int sw = __float_as_uint(v.w) >> 31;
                                if (sx) v.x *= p.slope; if (fabsf(v.x) > p.clamp) { sx = 2; v.x = InternalType<T>::clamp(v.x, p.clamp); }
                                if (sy) v.y *= p.slope; if (fabsf(v.y) > p.clamp) { sy = 2; v.y = InternalType<T>::clamp(v.y, p.clamp); }
                                if (sz) v.z *= p.slope; if (fabsf(v.z) > p.clamp) { sz = 2; v.z = InternalType<T>::clamp(v.z, p.clamp); }
                                if (sw) v.w *= p.slope; if (fabsf(v.w) > p.clamp) { sw = 2; v.w = InternalType<T>::clamp(v.w, p.clamp); }

                                p.s[si] = sx + (sy << 2) + (sz << 4) + (sw << 6);
                            }
                            else
                            {
                                // Just compute the values.
                                if (v.x < 0.f) v.x *= p.slope; v.x = InternalType<T>::clamp(v.x, p.clamp);
                                if (v.y < 0.f) v.y *= p.slope; v.y = InternalType<T>::clamp(v.y, p.clamp);
                                if (v.z < 0.f) v.z *= p.slope; v.z = InternalType<T>::clamp(v.z, p.clamp);
                                if (v.w < 0.f) v.w *= p.slope; v.w = InternalType<T>::clamp(v.w, p.clamp);
                            }
                        }
                    }
                    else if (signRead) // Read sign and apply.
                    {
                        if ((uint32_t)signY < p.sShape.y)
                        {
                            int s = 0;
                            if ((uint32_t)signXb     < p.swLimit) s  = p.s[si];
                            if ((uint32_t)signXb + 1 < p.swLimit) s |= p.s[si + 1] << 8;
                            s >>= (signX & 3) << 1;
                            if (s & 0x01) v.x *= p.slope; if (s & 0x02) v.x = 0.f;
                            if (s & 0x04) v.y *= p.slope; if (s & 0x08) v.y = 0.f;
                            if (s & 0x10) v.z *= p.slope; if (s & 0x20) v.z = 0.f;
                            if (s & 0x40) v.w *= p.slope; if (s & 0x80) v.w = 0.f;
                        }
                    }
                    else // Forward pass with no sign write.
                    {
                        if (v.x < 0.f) v.x *= p.slope; v.x = InternalType<T>::clamp(v.x, p.clamp);
                        if (v.y < 0.f) v.y *= p.slope; v.y = InternalType<T>::clamp(v.y, p.clamp);
                        if (v.z < 0.f) v.z *= p.slope; v.z = InternalType<T>::clamp(v.z, p.clamp);
                        if (v.w < 0.f) v.w *= p.slope; v.w = InternalType<T>::clamp(v.w, p.clamp);
                    }

                    s_tileUpXY[idx + 0] = v.x;
                    s_tileUpXY[idx + 1] = v.y;
                    s_tileUpXY[idx + 2] = v.z;
                    s_tileUpXY[idx + 3] = v.w;
                }
            }
            else if (up == 1)
            {
                __syncthreads();
                uint32_t groupMask = 15 << ((threadIdx.x & 31) & ~3);
                int minY = tileOutY ? (tileOutY - tileOutH) * down + tileUpH : 0; // Skip already written signs.
                for (int idx = threadIdx.x; idx < tileUpW * tileUpH; idx += blockDim.x)
                {
                    int relUpX0, relUpY0;
                    fast_div_mod<tileUpW>(relUpX0, relUpY0, idx);
                    scalar_t v = s_tileIn[idx] * (scalar_t)c_fu[0]; // 1x1 filter.

                    int x = tileOutX * down + relUpX0;
                    int y = tileOutY * down + relUpY0;
                    int signX = x + p.sOfs.x;
                    int signY = y + p.sOfs.y;
                    int signZ = blockIdx.z + p.blockZofs;
                    int signXb = signX >> 2;
                    index_t si = signXb + p.sShape.x * (signY + (index_t)p.sShape.y * signZ);
                    v *= (scalar_t)((float)up * (float)up * p.gain);

                    if (signWrite)
                    {
                        if (!enableWriteSkip)
                        {
                            // Determine and write sign.
                            uint32_t s = 0;
                            uint32_t signXbit = (1u << signXo);
                            if (v < 0.f)
                            {
                                s = signXbit;
                                v *= p.slope;
                            }
                            if (fabsf(v) > p.clamp)
                            {
                                s = signXbit * 2;
                                v = InternalType<T>::clamp(v, p.clamp);
                            }
                            if ((uint32_t)signXb < p.swLimit && (uint32_t)signY < p.sShape.y && signY >= minY)
                            {
                                s += __shfl_xor_sync(groupMask, s, 1);  // Coalesce.
                                s += __shfl_xor_sync(groupMask, s, 2);  // Coalesce.
                                p.s[si] = s;                            // Write.
                            }
                        }
                        else
                        {
                            // Determine and write sign.
                            if ((uint32_t)signXb < p.swLimit && (uint32_t)signY < p.sShape.y && signY >= minY)
                            {
                                uint32_t s = 0;
                                uint32_t signXbit = (1u << signXo);
                                if (v < 0.f)
                                {
                                    s = signXbit;
                                    v *= p.slope;
                                }
                                if (fabsf(v) > p.clamp)
                                {
                                    s = signXbit * 2;
                                    v = InternalType<T>::clamp(v, p.clamp);
                                }
                                s += __shfl_xor_sync(groupMask, s, 1);  // Coalesce.
                                s += __shfl_xor_sync(groupMask, s, 2);  // Coalesce.
                                p.s[si] = s;                            // Write.
                            }
                            else
                            {
                                // Just compute the value.
                                if (v < 0.f) v *= p.slope;
                                v = InternalType<T>::clamp(v, p.clamp);
                            }
                        }
                    }
                    else if (signRead)
                    {
                        // Read sign and apply if within sign tensor bounds.
                        if ((uint32_t)signXb < p.swLimit && (uint32_t)signY < p.sShape.y)
                        {
                            int s = p.s[si];
                            s >>= signXo;
                            if (s & 1) v *= p.slope;
                            if (s & 2) v = 0.f;
                        }
                    }
                    else // Forward pass with no sign write.
                    {
                        if (v < 0.f) v *= p.slope;
                        v = InternalType<T>::clamp(v, p.clamp);
                    }

                    if (!downInline) // Write into temporary buffer.
                        s_tileUpXY[idx] = v;
                    else if ((uint32_t)x < p.yShape.x && (uint32_t)y < p.yShape.y) // Write directly into output buffer
                        *((T*)((char*)p.y + (x * get_stride<index_t>(p.yStride.x) + y * get_stride<index_t>(p.yStride.y) + mapOfsOut))) = (T)(v * (scalar_t)c_fd[0]);
                }
            }
        }

        // Downsampling.
        if (filterMode == MODE_SUSD || filterMode == MODE_FUSD)
        {
            // Horizontal downsampling.
            __syncthreads();
            if (down == 4 && tileOutW % 4 == 0)
            {
                // Calculate 4 pixels at a time.
                for (int idx = threadIdx.x * 4; idx < tileOutW * tileUpH; idx += blockDim.x * 4)
                {
                    int relOutX0, relUpY;
                    fast_div_mod<tileOutW>(relOutX0, relUpY, idx);
                    int relUpX0 = relOutX0 * down;
                    int src0 = relUpY * tileUpW + relUpX0;
                    vec4_t v = InternalType<T>::zero_vec4();
                    #pragma unroll
                    for (int step = 0; step < fdSize; step++)
                    {
                        v.x += s_tileUpXY[src0 +  0 + step] * (scalar_t)c_fd[step];
                        v.y += s_tileUpXY[src0 +  4 + step] * (scalar_t)c_fd[step];
                        v.z += s_tileUpXY[src0 +  8 + step] * (scalar_t)c_fd[step];
                        v.w += s_tileUpXY[src0 + 12 + step] * (scalar_t)c_fd[step];
                    }
                    s_tileDownX[idx+0] = v.x;
                    s_tileDownX[idx+1] = v.y;
                    s_tileDownX[idx+2] = v.z;
                    s_tileDownX[idx+3] = v.w;
                }
            }
            else if ((down == 2 || down == 4) && (tileOutW % 2 == 0))
            {
                // Calculate 2 pixels at a time.
                for (int idx = threadIdx.x * 2; idx < tileOutW * tileUpH; idx += blockDim.x * 2)
                {
                    int relOutX0, relUpY;
                    fast_div_mod<tileOutW>(relOutX0, relUpY, idx);
                    int relUpX0 = relOutX0 * down;
                    int src0 = relUpY * tileUpW + relUpX0;
                    vec2_t v = InternalType<T>::zero_vec2();
                    #pragma unroll
                    for (int step = 0; step < fdSize; step++)
                    {
                        v.x += s_tileUpXY[src0 +    0 + step] * (scalar_t)c_fd[step];
                        v.y += s_tileUpXY[src0 + down + step] * (scalar_t)c_fd[step];
                    }
                    s_tileDownX[idx+0] = v.x;
                    s_tileDownX[idx+1] = v.y;
                }
            }
            else
            {
                // Calculate 1 pixel at a time.
                for (int idx = threadIdx.x; idx < tileOutW * tileUpH; idx += blockDim.x)
                {
                    int relOutX0, relUpY;
                    fast_div_mod<tileOutW>(relOutX0, relUpY, idx);
                    int relUpX0 = relOutX0 * down;
                    int src = relUpY * tileUpW + relUpX0;
                    scalar_t v = 0.f;
                    #pragma unroll
                    for (int step = 0; step < fdSize; step++)
                        v += s_tileUpXY[src + step] * (scalar_t)c_fd[step];
                    s_tileDownX[idx] = v;
                }
            }

            // Vertical downsampling & store output tile.
            __syncthreads();
            for (int idx = threadIdx.x; idx < tileOutW * tileOutH; idx += blockDim.x)
            {
                int relOutX, relOutY0;
                fast_div_mod<tileOutW>(relOutX, relOutY0, idx);
                int relUpY0 = relOutY0 * down;
                int src0 = relUpY0 * tileOutW + relOutX;
                scalar_t v = 0;
                #pragma unroll
                for (int step = 0; step < fdSize; step++)
                    v += s_tileDownX[src0 + step * tileOutW] * (scalar_t)c_fd[step];

                int outX = tileOutX + relOutX;
                int outY = tileOutY + relOutY0;

                if (outX < p.yShape.x & outY < p.yShape.y)
                    *((T*)((char*)p.y + (outX * get_stride<index_t>(p.yStride.x) + outY * get_stride<index_t>(p.yStride.y) + mapOfsOut))) = (T)v;
            }
        }
        else if (filterMode == MODE_SUFD || filterMode == MODE_FUFD)
        {
            // Full downsampling filter.
            if (down == 2)
            {
                // 2-wide.
                __syncthreads();
                for (int idx = threadIdx.x * 2; idx < tileOutW * tileOutH; idx += blockDim.x * 2)
                {
                    int relOutX0, relOutY0;
                    fast_div_mod<tileOutW>(relOutX0, relOutY0, idx);
                    int relUpX0 = relOutX0 * down;
                    int relUpY0 = relOutY0 * down;
                    int src0 = relUpY0 * tileUpW + relUpX0;
                    vec2_t v = InternalType<T>::zero_vec2();
                    #pragma unroll
                    for (int sy = 0; sy < fdSize; sy++)
                    #pragma unroll
                    for (int sx = 0; sx < fdSize; sx++)
                    {
                        v.x += s_tileUpXY[src0 + 0 + sx + sy * tileUpW] * (scalar_t)c_fd[sx + sy * MAX_FILTER_SIZE];
                        v.y += s_tileUpXY[src0 + 2 + sx + sy * tileUpW] * (scalar_t)c_fd[sx + sy * MAX_FILTER_SIZE];
                    }

                    int outX = tileOutX + relOutX0;
                    int outY = tileOutY + relOutY0;
                    if ((uint32_t)outY < p.yShape.y)
                    {
                        index_t ofs = outX * get_stride<index_t>(p.yStride.x) + outY * get_stride<index_t>(p.yStride.y) + mapOfsOut;
                        if (outX + 0 < p.yShape.x) *((T*)((char*)p.y + ofs)) = (T)v.x;
                        if (outX + 1 < p.yShape.x) *((T*)((char*)p.y + ofs + get_stride<index_t>(p.yStride.x))) = (T)v.y;
                    }
                }
            }
            else if (down == 1 && !downInline)
            {
                // Thread per pixel.
                __syncthreads();
                for (int idx = threadIdx.x; idx < tileOutW * tileOutH; idx += blockDim.x)
                {
                    int relOutX0, relOutY0;
                    fast_div_mod<tileOutW>(relOutX0, relOutY0, idx);
                    scalar_t v = s_tileUpXY[idx] * (scalar_t)c_fd[0]; // 1x1 filter.

                    int outX = tileOutX + relOutX0;
                    int outY = tileOutY + relOutY0;
                    if ((uint32_t)outX < p.yShape.x && (uint32_t)outY < p.yShape.y)
                        *((T*)((char*)p.y + (outX * get_stride<index_t>(p.yStride.x) + outY * get_stride<index_t>(p.yStride.y) + mapOfsOut))) = (T)v;
                }
            }
        }

        if (!enableXrep)
            break;
    }
}

//------------------------------------------------------------------------
// Compute activation function and signs for upsampled data tensor, modifying data tensor in-place. Used for accelerating the generic variant.
// Sign tensor is known to be contiguous, and p.x and p.s have the same z, w dimensions. 64-bit indexing is always used.

template <class T, bool signWrite, bool signRead>
static __global__ void filtered_lrelu_act_kernel(filtered_lrelu_act_kernel_params p)
{
    typedef typename InternalType<T>::scalar_t scalar_t;

    // Indexing.
    int32_t x = threadIdx.x + blockIdx.x * blockDim.x;
    int32_t ymax = signWrite ? p.sShape.y : p.xShape.y;
    int32_t qmax = p.xShape.z * p.xShape.w; // Combined minibatch*channel maximum index.

    // Loop to accommodate oversized tensors.
    for (int32_t q = blockIdx.z; q < qmax; q += gridDim.z)
    for (int32_t y = blockIdx.y; y < ymax; y += gridDim.y)
    {
        // Extract z and w (channel, minibatch index).
        int32_t w = q / p.xShape.z;
        int32_t z = q - w * p.xShape.z;

        // Choose behavior based on sign read/write mode.
        if (signWrite)
        {
            // Process value if in p.x.
            uint32_t s = 0;
            if (x < p.xShape.x && y < p.xShape.y)
            {
                int64_t ix = x * p.xStride.x + y * p.xStride.y + z * p.xStride.z + w * p.xStride.w;
                T* pv = ((T*)p.x) + ix;
                scalar_t v = (scalar_t)(*pv);

                // Gain, LReLU, clamp.
                v *= p.gain;
                if (v < 0.f)
                {
                    v *= p.slope;
                    s = 1; // Sign.
                }
                if (fabsf(v) > p.clamp)
                {
                    v = InternalType<T>::clamp(v, p.clamp);
                    s = 2; // Clamp.
                }

                *pv = (T)v; // Write value.
            }

            // Coalesce into threads 0 and 16 of warp.
            uint32_t m = (threadIdx.x & 16) ? 0xffff0000u : 0x0000ffffu;
            s <<= ((threadIdx.x & 15) << 1); // Shift into place.
            s |= __shfl_xor_sync(m, s, 1); // Distribute.
            s |= __shfl_xor_sync(m, s, 2);
            s |= __shfl_xor_sync(m, s, 4);
            s |= __shfl_xor_sync(m, s, 8);

            // Write signs if leader and in p.s.
            if (!(threadIdx.x & 15) && x < p.sShape.x) // y is always in.
            {
                uint64_t is = x + p.sShape.x * (y + (int64_t)p.sShape.y * q); // Contiguous.
                ((uint32_t*)p.s)[is >> 4] = s;
            }
        }
        else if (signRead)
        {
            // Process value if in p.x.
            if (x < p.xShape.x) // y is always in.
            {
                int64_t ix = x * p.xStride.x + y * p.xStride.y + z * p.xStride.z + w * p.xStride.w;
                T* pv = ((T*)p.x) + ix;
                scalar_t v = (scalar_t)(*pv);
                v *= p.gain;

                // Apply sign buffer offset.
                uint32_t sx = x + p.sOfs.x;
                uint32_t sy = y + p.sOfs.y;

                // Read and apply signs if we land inside valid region of sign buffer.
                if (sx < p.sShape.x && sy < p.sShape.y)
                {
                    uint64_t is = (sx >> 2) + (p.sShape.x >> 2) * (sy + (uint64_t)p.sShape.y * q); // Contiguous.
                    unsigned char s = p.s[is];
                    s >>= (sx & 3) << 1; // Shift into place.
                    if (s & 1) // Sign?
                        v *= p.slope;
                    if (s & 2) // Clamp?
                        v = 0.f;
                }

                *pv = (T)v; // Write value.
            }
        }
        else
        {
            // Forward pass with no sign write. Process value if in p.x.
            if (x < p.xShape.x) // y is always in.
            {
                int64_t ix = x * p.xStride.x + y * p.xStride.y + z * p.xStride.z + w * p.xStride.w;
                T* pv = ((T*)p.x) + ix;
                scalar_t v = (scalar_t)(*pv);
                v *= p.gain;
                if (v < 0.f)
                    v *= p.slope;
                if (fabsf(v) > p.clamp)
                    v = InternalType<T>::clamp(v, p.clamp);
                *pv = (T)v; // Write value.
            }
        }
    }
}

template <class T, bool signWrite, bool signRead> void* choose_filtered_lrelu_act_kernel(void)
{
    return (void*)filtered_lrelu_act_kernel<T, signWrite, signRead>;
}

//------------------------------------------------------------------------
// CUDA kernel selection.

template <class T, class index_t, bool signWrite, bool signRead> filtered_lrelu_kernel_spec choose_filtered_lrelu_kernel(const filtered_lrelu_kernel_params& p, int sharedKB)
{
    filtered_lrelu_kernel_spec s = { 0 };

    // Return the first matching kernel.
#define CASE(SH, U, FU, D, FD, MODE, TW, TH, W, XR, WS) \
    if (sharedKB >= SH) \
    if ((p.fuShape.y == 0 && (MODE == MODE_SUSD || MODE == MODE_SUFD)) || (p.fuShape.y > 0 && (MODE == MODE_FUSD || MODE == MODE_FUFD))) \
    if ((p.fdShape.y == 0 && (MODE == MODE_SUSD || MODE == MODE_FUSD)) || (p.fdShape.y > 0 && (MODE == MODE_SUFD || MODE == MODE_FUFD))) \
    if (p.up == U && p.fuShape.x <= FU && p.fuShape.y <= FU && p.down == D && p.fdShape.x <= FD && p.fdShape.y <= FD) \
    { \
        static_assert((D*TW % 4) == 0, "down * tileWidth must be divisible by 4"); \
        static_assert(FU % U == 0, "upscaling filter size must be multiple of upscaling factor"); \
        static_assert(FD % D == 0, "downscaling filter size must be multiple of downscaling factor"); \
        s.setup = (void*)setup_filters_kernel; \
        s.exec = (void*)filtered_lrelu_kernel<T, index_t, SH, signWrite, signRead, MODE, U, FU, D, FD, TW, TH, W*32, !!XR, !!WS>; \
        s.tileOut = make_int2(TW, TH); \
        s.numWarps = W; \
        s.xrep = XR; \
        s.dynamicSharedKB = (SH == 48) ? 0 : SH; \
        return s; \
    }

    // Launch parameters for various kernel specializations.
    // Small filters must be listed before large filters, otherwise the kernel for larger filter will always match first.
    // Kernels that use more shared memory must be listed before those that use less, for the same reason.

    CASE(/*sharedKB*/48, /*up,fu*/1,1,  /*down,fd*/1,1,  /*mode*/MODE_FUFD, /*tw,th,warps,xrep,wskip*/64,  178, 32,  0,  0) // 1t-upf1-downf1
    CASE(/*sharedKB*/48, /*up,fu*/2,8,  /*down,fd*/1,1,  /*mode*/MODE_SUFD, /*tw,th,warps,xrep,wskip*/152, 95,  16,  0,  0) // 4t-ups2-downf1
    CASE(/*sharedKB*/48, /*up,fu*/1,1,  /*down,fd*/2,8,  /*mode*/MODE_FUSD, /*tw,th,warps,xrep,wskip*/56,  22,  16,  0,  0) // 4t-upf1-downs2
    CASE(/*sharedKB*/48, /*up,fu*/2,8,  /*down,fd*/2,8,  /*mode*/MODE_SUSD, /*tw,th,warps,xrep,wskip*/56,  29,  16,  11, 0) // 4t-ups2-downs2
    CASE(/*sharedKB*/48, /*up,fu*/2,8,  /*down,fd*/2,8,  /*mode*/MODE_FUSD, /*tw,th,warps,xrep,wskip*/60,  28,  16,  0,  0) // 4t-upf2-downs2
    CASE(/*sharedKB*/48, /*up,fu*/2,8,  /*down,fd*/2,8,  /*mode*/MODE_SUFD, /*tw,th,warps,xrep,wskip*/56,  28,  16,  0,  0) // 4t-ups2-downf2
    CASE(/*sharedKB*/48, /*up,fu*/4,16, /*down,fd*/2,8,  /*mode*/MODE_SUSD, /*tw,th,warps,xrep,wskip*/56,  31,  16,  11, 0) // 4t-ups4-downs2
    CASE(/*sharedKB*/48, /*up,fu*/4,16, /*down,fd*/2,8,  /*mode*/MODE_SUFD, /*tw,th,warps,xrep,wskip*/56,  36,  16,  0,  0) // 4t-ups4-downf2
    CASE(/*sharedKB*/48, /*up,fu*/2,8,  /*down,fd*/4,16, /*mode*/MODE_SUSD, /*tw,th,warps,xrep,wskip*/16,  22,  16,  12, 0) // 4t-ups2-downs4
    CASE(/*sharedKB*/48, /*up,fu*/2,8,  /*down,fd*/4,16, /*mode*/MODE_FUSD, /*tw,th,warps,xrep,wskip*/29,  15,  16,  0,  0) // 4t-upf2-downs4
    CASE(/*sharedKB*/48, /*up,fu*/2,12, /*down,fd*/1,1,  /*mode*/MODE_SUFD, /*tw,th,warps,xrep,wskip*/96,  150, 28,  0,  0) // 6t-ups2-downf1
    CASE(/*sharedKB*/48, /*up,fu*/1,1,  /*down,fd*/2,12, /*mode*/MODE_FUSD, /*tw,th,warps,xrep,wskip*/32,  35,  24,  0,  0) // 6t-upf1-downs2
    CASE(/*sharedKB*/48, /*up,fu*/2,12, /*down,fd*/2,12, /*mode*/MODE_SUSD, /*tw,th,warps,xrep,wskip*/32,  46,  16,  10, 0) // 6t-ups2-downs2
    CASE(/*sharedKB*/48, /*up,fu*/2,12, /*down,fd*/2,12, /*mode*/MODE_FUSD, /*tw,th,warps,xrep,wskip*/58,  28,  24,  8,  0) // 6t-upf2-downs2
    CASE(/*sharedKB*/48, /*up,fu*/2,12, /*down,fd*/2,12, /*mode*/MODE_SUFD, /*tw,th,warps,xrep,wskip*/52,  28,  16,  0,  0) // 6t-ups2-downf2
    CASE(/*sharedKB*/48, /*up,fu*/4,24, /*down,fd*/2,12, /*mode*/MODE_SUSD, /*tw,th,warps,xrep,wskip*/32,  51,  16,  5,  0) // 6t-ups4-downs2
    CASE(/*sharedKB*/48, /*up,fu*/4,24, /*down,fd*/2,12, /*mode*/MODE_SUFD, /*tw,th,warps,xrep,wskip*/32,  56,  16,  6,  0) // 6t-ups4-downf2
    CASE(/*sharedKB*/48, /*up,fu*/2,12, /*down,fd*/4,24, /*mode*/MODE_SUSD, /*tw,th,warps,xrep,wskip*/16,  18,  16,  12, 0) // 6t-ups2-downs4
    CASE(/*sharedKB*/96, /*up,fu*/2,12, /*down,fd*/4,24, /*mode*/MODE_FUSD, /*tw,th,warps,xrep,wskip*/27,  31,  32,  6,  0) // 6t-upf2-downs4 96kB
    CASE(/*sharedKB*/48, /*up,fu*/2,12, /*down,fd*/4,24, /*mode*/MODE_FUSD, /*tw,th,warps,xrep,wskip*/27,  13,  24,  0,  0) // 6t-upf2-downs4
    CASE(/*sharedKB*/48, /*up,fu*/2,16, /*down,fd*/1,1,  /*mode*/MODE_SUFD, /*tw,th,warps,xrep,wskip*/148, 89,  24,  0,  0) // 8t-ups2-downf1
    CASE(/*sharedKB*/48, /*up,fu*/1,1,  /*down,fd*/2,16, /*mode*/MODE_FUSD, /*tw,th,warps,xrep,wskip*/32,  31,  16,  5,  0) // 8t-upf1-downs2
    CASE(/*sharedKB*/48, /*up,fu*/2,16, /*down,fd*/2,16, /*mode*/MODE_SUSD, /*tw,th,warps,xrep,wskip*/32,  41,  16,  9,  0) // 8t-ups2-downs2
    CASE(/*sharedKB*/48, /*up,fu*/2,16, /*down,fd*/2,16, /*mode*/MODE_FUSD, /*tw,th,warps,xrep,wskip*/56,  26,  24,  0,  0) // 8t-upf2-downs2
    CASE(/*sharedKB*/48, /*up,fu*/2,16, /*down,fd*/2,16, /*mode*/MODE_SUFD, /*tw,th,warps,xrep,wskip*/32,  40,  16,  0,  0) // 8t-ups2-downf2
    CASE(/*sharedKB*/48, /*up,fu*/4,32, /*down,fd*/2,16, /*mode*/MODE_SUSD, /*tw,th,warps,xrep,wskip*/32,  46,  24,  5,  0) // 8t-ups4-downs2
    CASE(/*sharedKB*/48, /*up,fu*/4,32, /*down,fd*/2,16, /*mode*/MODE_SUFD, /*tw,th,warps,xrep,wskip*/32,  50,  16,  0,  0) // 8t-ups4-downf2
    CASE(/*sharedKB*/96, /*up,fu*/2,16, /*down,fd*/4,32, /*mode*/MODE_SUSD, /*tw,th,warps,xrep,wskip*/24,  24,  32,  12, 1) // 8t-ups2-downs4 96kB
    CASE(/*sharedKB*/48, /*up,fu*/2,16, /*down,fd*/4,32, /*mode*/MODE_SUSD, /*tw,th,warps,xrep,wskip*/16,  13,  16,  10, 1) // 8t-ups2-downs4
    CASE(/*sharedKB*/96, /*up,fu*/2,16, /*down,fd*/4,32, /*mode*/MODE_FUSD, /*tw,th,warps,xrep,wskip*/25,  28,  28,  4,  0) // 8t-upf2-downs4 96kB
    CASE(/*sharedKB*/48, /*up,fu*/2,16, /*down,fd*/4,32, /*mode*/MODE_FUSD, /*tw,th,warps,xrep,wskip*/25,  10,  24,  0,  0) // 8t-upf2-downs4

    #undef CASE
    return s; // No kernel found.
}

//------------------------------------------------------------------------
