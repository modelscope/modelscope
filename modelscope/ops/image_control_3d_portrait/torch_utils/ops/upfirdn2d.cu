#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2021-2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: LicenseRef-NvidiaProprietary
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
 */

#include <c10/util/Half.h>
#include "upfirdn2d.h"

//------------------------------------------------------------------------
// Helpers.

template <class T> struct InternalType;
template <> struct InternalType<double>     { typedef double scalar_t; };
template <> struct InternalType<float>      { typedef float  scalar_t; };
template <> struct InternalType<c10::Half>  { typedef float  scalar_t; };

static __device__ __forceinline__ int floor_div(int a, int b)
{
    int t = 1 - a / b;
    return (a + t * b) / b - t;
}

//------------------------------------------------------------------------
// Generic CUDA implementation for large filters.

template <class T> static __global__ void upfirdn2d_kernel_large(upfirdn2d_kernel_params p)
{
    typedef typename InternalType<T>::scalar_t scalar_t;

    // Calculate thread index.
    int minorBase = blockIdx.x * blockDim.x + threadIdx.x;
    int outY = minorBase / p.launchMinor;
    minorBase -= outY * p.launchMinor;
    int outXBase = blockIdx.y * p.loopX * blockDim.y + threadIdx.y;
    int majorBase = blockIdx.z * p.loopMajor;
    if (outXBase >= p.outSize.x | outY >= p.outSize.y | majorBase >= p.sizeMajor)
        return;

    // Setup Y receptive field.
    int midY = outY * p.down.y + p.up.y - 1 - p.pad0.y;
    int inY = min(max(floor_div(midY, p.up.y), 0), p.inSize.y);
    int h = min(max(floor_div(midY + p.filterSize.y, p.up.y), 0), p.inSize.y) - inY;
    int filterY = midY + p.filterSize.y - (inY + 1) * p.up.y;
    if (p.flip)
        filterY = p.filterSize.y - 1 - filterY;

    // Loop over major, minor, and X.
    for (int majorIdx = 0, major = majorBase; majorIdx < p.loopMajor & major < p.sizeMajor; majorIdx++, major++)
    for (int minorIdx = 0, minor = minorBase; minorIdx < p.loopMinor & minor < p.sizeMinor; minorIdx++, minor += p.launchMinor)
    {
        int nc = major * p.sizeMinor + minor;
        int n = nc / p.inSize.z;
        int c = nc - n * p.inSize.z;
        for (int loopX = 0, outX = outXBase; loopX < p.loopX & outX < p.outSize.x; loopX++, outX += blockDim.y)
        {
            // Setup X receptive field.
            int midX = outX * p.down.x + p.up.x - 1 - p.pad0.x;
            int inX = min(max(floor_div(midX, p.up.x), 0), p.inSize.x);
            int w = min(max(floor_div(midX + p.filterSize.x, p.up.x), 0), p.inSize.x) - inX;
            int filterX = midX + p.filterSize.x - (inX + 1) * p.up.x;
            if (p.flip)
                filterX = p.filterSize.x - 1 - filterX;

            // Initialize pointers.
            const T* xp = &((const T*)p.x)[inX * p.inStride.x + inY * p.inStride.y + c * p.inStride.z + n * p.inStride.w];
            const float* fp = &p.f[filterX * p.filterStride.x + filterY * p.filterStride.y];
            int filterStepX = ((p.flip) ? p.up.x : -p.up.x) * p.filterStride.x;
            int filterStepY = ((p.flip) ? p.up.y : -p.up.y) * p.filterStride.y;

            // Inner loop.
            scalar_t v = 0;
            for (int y = 0; y < h; y++)
            {
                for (int x = 0; x < w; x++)
                {
                    v += (scalar_t)(*xp) * (scalar_t)(*fp);
                    xp += p.inStride.x;
                    fp += filterStepX;
                }
                xp += p.inStride.y - w * p.inStride.x;
                fp += filterStepY - w * filterStepX;
            }

            // Store result.
            v *= p.gain;
            ((T*)p.y)[outX * p.outStride.x + outY * p.outStride.y + c * p.outStride.z + n * p.outStride.w] = (T)v;
        }
    }
}

//------------------------------------------------------------------------
// Specialized CUDA implementation for small filters.

template <class T, int upx, int upy, int downx, int downy, int filterW, int filterH, int tileOutW, int tileOutH, int loopMinor>
static __global__ void upfirdn2d_kernel_small(upfirdn2d_kernel_params p)
{
    typedef typename InternalType<T>::scalar_t scalar_t;
    const int tileInW = ((tileOutW - 1) * downx + filterW - 1) / upx + 1;
    const int tileInH = ((tileOutH - 1) * downy + filterH - 1) / upy + 1;
    __shared__ volatile scalar_t sf[filterH][filterW];
    __shared__ volatile scalar_t sx[tileInH][tileInW][loopMinor];

    // Calculate tile index.
    int minorBase = blockIdx.x;
    int tileOutY = minorBase / p.launchMinor;
    minorBase -= tileOutY * p.launchMinor;
    minorBase *= loopMinor;
    tileOutY *= tileOutH;
    int tileOutXBase = blockIdx.y * p.loopX * tileOutW;
    int majorBase = blockIdx.z * p.loopMajor;
    if (tileOutXBase >= p.outSize.x | tileOutY >= p.outSize.y | majorBase >= p.sizeMajor)
        return;

    // Load filter (flipped).
    for (int tapIdx = threadIdx.x; tapIdx < filterH * filterW; tapIdx += blockDim.x)
    {
        int fy = tapIdx / filterW;
        int fx = tapIdx - fy * filterW;
        scalar_t v = 0;
        if (fx < p.filterSize.x & fy < p.filterSize.y)
        {
            int ffx = (p.flip) ? fx : p.filterSize.x - 1 - fx;
            int ffy = (p.flip) ? fy : p.filterSize.y - 1 - fy;
            v = (scalar_t)p.f[ffx * p.filterStride.x + ffy * p.filterStride.y];
        }
        sf[fy][fx] = v;
    }

    // Loop over major and X.
    for (int majorIdx = 0, major = majorBase; majorIdx < p.loopMajor & major < p.sizeMajor; majorIdx++, major++)
    {
        int baseNC = major * p.sizeMinor + minorBase;
        int n = baseNC / p.inSize.z;
        int baseC = baseNC - n * p.inSize.z;
        for (int loopX = 0, tileOutX = tileOutXBase; loopX < p.loopX & tileOutX < p.outSize.x; loopX++, tileOutX += tileOutW)
        {
            // Load input pixels.
            int tileMidX = tileOutX * downx + upx - 1 - p.pad0.x;
            int tileMidY = tileOutY * downy + upy - 1 - p.pad0.y;
            int tileInX = floor_div(tileMidX, upx);
            int tileInY = floor_div(tileMidY, upy);
            __syncthreads();
            for (int inIdx = threadIdx.x; inIdx < tileInH * tileInW * loopMinor; inIdx += blockDim.x)
            {
                int relC = inIdx;
                int relInX = relC / loopMinor;
                int relInY = relInX / tileInW;
                relC -= relInX * loopMinor;
                relInX -= relInY * tileInW;
                int c = baseC + relC;
                int inX = tileInX + relInX;
                int inY = tileInY + relInY;
                scalar_t v = 0;
                if (inX >= 0 & inY >= 0 & inX < p.inSize.x & inY < p.inSize.y & c < p.inSize.z)
                    v = (scalar_t)((const T*)p.x)[inX * p.inStride.x + inY * p.inStride.y + c * p.inStride.z + n * p.inStride.w];
                sx[relInY][relInX][relC] = v;
            }

            // Loop over output pixels.
            __syncthreads();
            for (int outIdx = threadIdx.x; outIdx < tileOutH * tileOutW * loopMinor; outIdx += blockDim.x)
            {
                int relC = outIdx;
                int relOutX = relC / loopMinor;
                int relOutY = relOutX / tileOutW;
                relC -= relOutX * loopMinor;
                relOutX -= relOutY * tileOutW;
                int c = baseC + relC;
                int outX = tileOutX + relOutX;
                int outY = tileOutY + relOutY;

                // Setup receptive field.
                int midX = tileMidX + relOutX * downx;
                int midY = tileMidY + relOutY * downy;
                int inX = floor_div(midX, upx);
                int inY = floor_div(midY, upy);
                int relInX = inX - tileInX;
                int relInY = inY - tileInY;
                int filterX = (inX + 1) * upx - midX - 1; // flipped
                int filterY = (inY + 1) * upy - midY - 1; // flipped

                // Inner loop.
                if (outX < p.outSize.x & outY < p.outSize.y & c < p.outSize.z)
                {
                    scalar_t v = 0;
                    #pragma unroll
                    for (int y = 0; y < filterH / upy; y++)
                        #pragma unroll
                        for (int x = 0; x < filterW / upx; x++)
                            v += sx[relInY + y][relInX + x][relC] * sf[filterY + y * upy][filterX + x * upx];
                    v *= p.gain;
                    ((T*)p.y)[outX * p.outStride.x + outY * p.outStride.y + c * p.outStride.z + n * p.outStride.w] = (T)v;
                }
            }
        }
    }
}

//------------------------------------------------------------------------
// CUDA kernel selection.

template <class T> upfirdn2d_kernel_spec choose_upfirdn2d_kernel(const upfirdn2d_kernel_params& p)
{
    int s = p.inStride.z, fx = p.filterSize.x, fy = p.filterSize.y;
    upfirdn2d_kernel_spec spec = {(void*)upfirdn2d_kernel_large<T>, -1,-1,1, 4}; // contiguous
    if (s == 1)           spec = {(void*)upfirdn2d_kernel_large<T>, -1,-1,4, 1}; // channels_last

    // No up/downsampling.
    if (p.up.x == 1 && p.up.y == 1 && p.down.x == 1 && p.down.y == 1)
    {
        // contiguous
        if (s != 1 && fx <= 24 && fy <= 24) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 24,24, 64,32,1>, 64,32,1, 1};
        if (s != 1 && fx <= 16 && fy <= 16) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 16,16, 64,32,1>, 64,32,1, 1};
        if (s != 1 && fx <= 7  && fy <= 7 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 7,7,   64,16,1>, 64,16,1, 1};
        if (s != 1 && fx <= 6  && fy <= 6 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 6,6,   64,16,1>, 64,16,1, 1};
        if (s != 1 && fx <= 5  && fy <= 5 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 5,5,   64,16,1>, 64,16,1, 1};
        if (s != 1 && fx <= 4  && fy <= 4 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 4,4,   64,16,1>, 64,16,1, 1};
        if (s != 1 && fx <= 3  && fy <= 3 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 3,3,   64,16,1>, 64,16,1, 1};
        if (s != 1 && fx <= 24 && fy <= 1 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 24,1,  128,8,1>, 128,8,1, 1};
        if (s != 1 && fx <= 16 && fy <= 1 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 16,1,  128,8,1>, 128,8,1, 1};
        if (s != 1 && fx <= 8  && fy <= 1 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 8,1,   128,8,1>, 128,8,1, 1};
        if (s != 1 && fx <= 1  && fy <= 24) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 1,24,  32,32,1>, 32,32,1, 1};
        if (s != 1 && fx <= 1  && fy <= 16) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 1,16,  32,32,1>, 32,32,1, 1};
        if (s != 1 && fx <= 1  && fy <= 8 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 1,8,   32,32,1>, 32,32,1, 1};
        // channels_last
        if (s == 1 && fx <= 24 && fy <= 24) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 24,24, 32,32,1>,  32,32,1,  1};
        if (s == 1 && fx <= 16 && fy <= 16) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 16,16, 32,32,1>,  32,32,1,  1};
        if (s == 1 && fx <= 7  && fy <= 7 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 7,7,   16,16,8>,  16,16,8,  1};
        if (s == 1 && fx <= 6  && fy <= 6 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 6,6,   16,16,8>,  16,16,8,  1};
        if (s == 1 && fx <= 5  && fy <= 5 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 5,5,   16,16,8>,  16,16,8,  1};
        if (s == 1 && fx <= 4  && fy <= 4 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 4,4,   16,16,8>,  16,16,8,  1};
        if (s == 1 && fx <= 3  && fy <= 3 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 3,3,   16,16,8>,  16,16,8,  1};
        if (s == 1 && fx <= 24 && fy <= 1 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 24,1,  128,1,16>, 128,1,16, 1};
        if (s == 1 && fx <= 16 && fy <= 1 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 16,1,  128,1,16>, 128,1,16, 1};
        if (s == 1 && fx <= 8  && fy <= 1 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 8,1,   128,1,16>, 128,1,16, 1};
        if (s == 1 && fx <= 1  && fy <= 24) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 1,24,  1,128,16>, 1,128,16, 1};
        if (s == 1 && fx <= 1  && fy <= 16) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 1,16,  1,128,16>, 1,128,16, 1};
        if (s == 1 && fx <= 1  && fy <= 8 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,1, 1,8,   1,128,16>, 1,128,16, 1};
    }

    // 2x upsampling.
    if (p.up.x == 2 && p.up.y == 2 && p.down.x == 1 && p.down.y == 1)
    {
        // contiguous
        if (s != 1 && fx <= 24 && fy <= 24) spec = {(void*)upfirdn2d_kernel_small<T, 2,2, 1,1, 24,24, 64,32,1>, 64,32,1, 1};
        if (s != 1 && fx <= 16 && fy <= 16) spec = {(void*)upfirdn2d_kernel_small<T, 2,2, 1,1, 16,16, 64,32,1>, 64,32,1, 1};
        if (s != 1 && fx <= 8  && fy <= 8 ) spec = {(void*)upfirdn2d_kernel_small<T, 2,2, 1,1, 8,8,   64,16,1>, 64,16,1, 1};
        if (s != 1 && fx <= 6  && fy <= 6 ) spec = {(void*)upfirdn2d_kernel_small<T, 2,2, 1,1, 6,6,   64,16,1>, 64,16,1, 1};
        if (s != 1 && fx <= 4  && fy <= 4 ) spec = {(void*)upfirdn2d_kernel_small<T, 2,2, 1,1, 4,4,   64,16,1>, 64,16,1, 1};
        if (s != 1 && fx <= 2  && fy <= 2 ) spec = {(void*)upfirdn2d_kernel_small<T, 2,2, 1,1, 2,2,   64,16,1>, 64,16,1, 1};
        // channels_last
        if (s == 1 && fx <= 24 && fy <= 24) spec = {(void*)upfirdn2d_kernel_small<T, 2,2, 1,1, 24,24, 32,32,1>, 32,32,1, 1};
        if (s == 1 && fx <= 16 && fy <= 16) spec = {(void*)upfirdn2d_kernel_small<T, 2,2, 1,1, 16,16, 32,32,1>, 32,32,1, 1};
        if (s == 1 && fx <= 8  && fy <= 8 ) spec = {(void*)upfirdn2d_kernel_small<T, 2,2, 1,1, 8,8,   16,16,8>, 16,16,8, 1};
        if (s == 1 && fx <= 6  && fy <= 6 ) spec = {(void*)upfirdn2d_kernel_small<T, 2,2, 1,1, 6,6,   16,16,8>, 16,16,8, 1};
        if (s == 1 && fx <= 4  && fy <= 4 ) spec = {(void*)upfirdn2d_kernel_small<T, 2,2, 1,1, 4,4,   16,16,8>, 16,16,8, 1};
        if (s == 1 && fx <= 2  && fy <= 2 ) spec = {(void*)upfirdn2d_kernel_small<T, 2,2, 1,1, 2,2,   16,16,8>, 16,16,8, 1};
    }
    if (p.up.x == 2 && p.up.y == 1 && p.down.x == 1 && p.down.y == 1)
    {
        // contiguous
        if (s != 1 && fx <= 24 && fy <= 1) spec = {(void*)upfirdn2d_kernel_small<T, 2,1, 1,1, 24,1, 128,8,1>, 128,8,1, 1};
        if (s != 1 && fx <= 16 && fy <= 1) spec = {(void*)upfirdn2d_kernel_small<T, 2,1, 1,1, 16,1, 128,8,1>, 128,8,1, 1};
        if (s != 1 && fx <= 8  && fy <= 1) spec = {(void*)upfirdn2d_kernel_small<T, 2,1, 1,1, 8,1,  128,8,1>, 128,8,1, 1};
        // channels_last
        if (s == 1 && fx <= 24 && fy <= 1) spec = {(void*)upfirdn2d_kernel_small<T, 2,1, 1,1, 24,1, 128,1,16>, 128,1,16, 1};
        if (s == 1 && fx <= 16 && fy <= 1) spec = {(void*)upfirdn2d_kernel_small<T, 2,1, 1,1, 16,1, 128,1,16>, 128,1,16, 1};
        if (s == 1 && fx <= 8  && fy <= 1) spec = {(void*)upfirdn2d_kernel_small<T, 2,1, 1,1, 8,1,  128,1,16>, 128,1,16, 1};
    }
    if (p.up.x == 1 && p.up.y == 2 && p.down.x == 1 && p.down.y == 1)
    {
        // contiguous
        if (s != 1 && fx <= 1 && fy <= 24) spec = {(void*)upfirdn2d_kernel_small<T, 1,2, 1,1, 1,24, 32,32,1>, 32,32,1, 1};
        if (s != 1 && fx <= 1 && fy <= 16) spec = {(void*)upfirdn2d_kernel_small<T, 1,2, 1,1, 1,16, 32,32,1>, 32,32,1, 1};
        if (s != 1 && fx <= 1 && fy <= 8 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,2, 1,1, 1,8,  32,32,1>, 32,32,1, 1};
        // channels_last
        if (s == 1 && fx <= 1 && fy <= 24) spec = {(void*)upfirdn2d_kernel_small<T, 1,2, 1,1, 1,24, 1,128,16>, 1,128,16, 1};
        if (s == 1 && fx <= 1 && fy <= 16) spec = {(void*)upfirdn2d_kernel_small<T, 1,2, 1,1, 1,16, 1,128,16>, 1,128,16, 1};
        if (s == 1 && fx <= 1 && fy <= 8 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,2, 1,1, 1,8,  1,128,16>, 1,128,16, 1};
    }

    // 2x downsampling.
    if (p.up.x == 1 && p.up.y == 1 && p.down.x == 2 && p.down.y == 2)
    {
        // contiguous
        if (s != 1 && fx <= 24 && fy <= 24) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 2,2, 24,24, 32,16,1>, 32,16,1, 1};
        if (s != 1 && fx <= 16 && fy <= 16) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 2,2, 16,16, 32,16,1>, 32,16,1, 1};
        if (s != 1 && fx <= 8  && fy <= 8 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 2,2, 8,8,   32,8,1>,  32,8,1,  1};
        if (s != 1 && fx <= 6  && fy <= 6 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 2,2, 6,6,   32,8,1>,  32,8,1,  1};
        if (s != 1 && fx <= 4  && fy <= 4 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 2,2, 4,4,   32,8,1>,  32,8,1,  1};
        if (s != 1 && fx <= 2  && fy <= 2 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 2,2, 2,2,   32,8,1>,  32,8,1,  1};
        // channels_last
        if (s == 1 && fx <= 24 && fy <= 24) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 2,2, 24,24, 16,16,1>, 16,16,1, 1};
        if (s == 1 && fx <= 16 && fy <= 16) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 2,2, 16,16, 16,16,1>, 16,16,1, 1};
        if (s == 1 && fx <= 8  && fy <= 8 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 2,2, 8,8,   8,8,8>,   8,8,8,   1};
        if (s == 1 && fx <= 6  && fy <= 6 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 2,2, 6,6,   8,8,8>,   8,8,8,   1};
        if (s == 1 && fx <= 4  && fy <= 4 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 2,2, 4,4,   8,8,8>,   8,8,8,   1};
        if (s == 1 && fx <= 2  && fy <= 2 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 2,2, 2,2,   8,8,8>,   8,8,8,   1};
    }
    if (p.up.x == 1 && p.up.y == 1 && p.down.x == 2 && p.down.y == 1)
    {
        // contiguous
        if (s != 1 && fx <= 24 && fy <= 1) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 2,1, 24,1, 64,8,1>, 64,8,1, 1};
        if (s != 1 && fx <= 16 && fy <= 1) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 2,1, 16,1, 64,8,1>, 64,8,1, 1};
        if (s != 1 && fx <= 8  && fy <= 1) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 2,1, 8,1,  64,8,1>, 64,8,1, 1};
        // channels_last
        if (s == 1 && fx <= 24 && fy <= 1) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 2,1, 24,1, 64,1,8>, 64,1,8, 1};
        if (s == 1 && fx <= 16 && fy <= 1) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 2,1, 16,1, 64,1,8>, 64,1,8, 1};
        if (s == 1 && fx <= 8  && fy <= 1) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 2,1, 8,1,  64,1,8>, 64,1,8, 1};
    }
    if (p.up.x == 1 && p.up.y == 1 && p.down.x == 1 && p.down.y == 2)
    {
        // contiguous
        if (s != 1 && fx <= 1 && fy <= 24) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,2, 1,24, 32,16,1>, 32,16,1, 1};
        if (s != 1 && fx <= 1 && fy <= 16) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,2, 1,16, 32,16,1>, 32,16,1, 1};
        if (s != 1 && fx <= 1 && fy <= 8 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,2, 1,8,  32,16,1>, 32,16,1, 1};
        // channels_last
        if (s == 1 && fx <= 1  && fy <= 24) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,2, 1,24, 1,64,8>, 1,64,8, 1};
        if (s == 1 && fx <= 1  && fy <= 16) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,2, 1,16, 1,64,8>, 1,64,8, 1};
        if (s == 1 && fx <= 1  && fy <= 8 ) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,2, 1,8,  1,64,8>, 1,64,8, 1};
    }

    // 4x upsampling.
    if (p.up.x == 4 && p.up.y == 4 && p.down.x == 1 && p.down.y == 1)
    {
        // contiguous
        if (s != 1 && fx <= 48 && fy <= 48) spec = {(void*)upfirdn2d_kernel_small<T, 4,4, 1,1, 48,48, 64,32,1>, 64,32,1, 1};
        if (s != 1 && fx <= 32 && fy <= 32) spec = {(void*)upfirdn2d_kernel_small<T, 4,4, 1,1, 32,32, 64,32,1>, 64,32,1, 1};
        // channels_last
        if (s == 1 && fx <= 48 && fy <= 48) spec = {(void*)upfirdn2d_kernel_small<T, 4,4, 1,1, 48,48, 32,32,1>, 32,32,1, 1};
        if (s == 1 && fx <= 32 && fy <= 32) spec = {(void*)upfirdn2d_kernel_small<T, 4,4, 1,1, 32,32, 32,32,1>, 32,32,1, 1};
    }
    if (p.up.x == 4 && p.up.y == 1 && p.down.x == 1 && p.down.y == 1)
    {
        // contiguous
        if (s != 1 && fx <= 48 && fy <= 1) spec = {(void*)upfirdn2d_kernel_small<T, 4,1, 1,1, 48,1, 128,8,1>, 128,8,1, 1};
        if (s != 1 && fx <= 32 && fy <= 1) spec = {(void*)upfirdn2d_kernel_small<T, 4,1, 1,1, 32,1, 128,8,1>, 128,8,1, 1};
        // channels_last
        if (s == 1 && fx <= 48 && fy <= 1) spec = {(void*)upfirdn2d_kernel_small<T, 4,1, 1,1, 48,1, 128,1,16>, 128,1,16, 1};
        if (s == 1 && fx <= 32 && fy <= 1) spec = {(void*)upfirdn2d_kernel_small<T, 4,1, 1,1, 32,1, 128,1,16>, 128,1,16, 1};
    }
    if (p.up.x == 1 && p.up.y == 4 && p.down.x == 1 && p.down.y == 1)
    {
        // contiguous
        if (s != 1 && fx <= 1 && fy <= 48) spec = {(void*)upfirdn2d_kernel_small<T, 1,4, 1,1, 1,48, 32,32,1>, 32,32,1, 1};
        if (s != 1 && fx <= 1 && fy <= 32) spec = {(void*)upfirdn2d_kernel_small<T, 1,4, 1,1, 1,32, 32,32,1>, 32,32,1, 1};
        // channels_last
        if (s == 1 && fx <= 1 && fy <= 48) spec = {(void*)upfirdn2d_kernel_small<T, 1,4, 1,1, 1,48, 1,128,16>, 1,128,16, 1};
        if (s == 1 && fx <= 1 && fy <= 32) spec = {(void*)upfirdn2d_kernel_small<T, 1,4, 1,1, 1,32, 1,128,16>, 1,128,16, 1};
    }

    // 4x downsampling (inefficient).
    if (p.up.x == 1 && p.up.y == 1 && p.down.x == 4 && p.down.y == 1)
    {
        // contiguous
        if (s != 1 && fx <= 48 && fy <= 1) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 4,1, 48,1, 32,8,1>, 32,8,1, 1};
        if (s != 1 && fx <= 32 && fy <= 1) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 4,1, 32,1, 32,8,1>, 32,8,1, 1};
        // channels_last
        if (s == 1 && fx <= 48 && fy <= 1) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 4,1, 48,1, 32,1,8>, 32,1,8, 1};
        if (s == 1 && fx <= 32 && fy <= 1) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 4,1, 32,1, 32,1,8>, 32,1,8, 1};
    }
    if (p.up.x == 1 && p.up.y == 1 && p.down.x == 1 && p.down.y == 4)
    {
        // contiguous
        if (s != 1 && fx <= 1 && fy <= 48) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,4, 1,48, 32,8,1>, 32,8,1, 1};
        if (s != 1 && fx <= 1 && fy <= 32) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,4, 1,32, 32,8,1>, 32,8,1, 1};
        // channels_last
        if (s == 1 && fx <= 1  && fy <= 48) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,4, 1,48, 1,32,8>, 1,32,8, 1};
        if (s == 1 && fx <= 1  && fy <= 32) spec = {(void*)upfirdn2d_kernel_small<T, 1,1, 1,4, 1,32, 1,32,8>, 1,32,8, 1};
    }
    return spec;
}

//------------------------------------------------------------------------
// Template specializations.

template upfirdn2d_kernel_spec choose_upfirdn2d_kernel<double>   (const upfirdn2d_kernel_params& p);
template upfirdn2d_kernel_spec choose_upfirdn2d_kernel<float>    (const upfirdn2d_kernel_params& p);
template upfirdn2d_kernel_spec choose_upfirdn2d_kernel<c10::Half>(const upfirdn2d_kernel_params& p);

//------------------------------------------------------------------------
