#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2021-2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: LicenseRef-NvidiaProprietary
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
 */

#include <c10/util/Half.h>
#include "bias_act.h"

//------------------------------------------------------------------------
// Helpers.

template <class T> struct InternalType;
template <> struct InternalType<double>     { typedef double scalar_t; };
template <> struct InternalType<float>      { typedef float  scalar_t; };
template <> struct InternalType<c10::Half>  { typedef float  scalar_t; };

//------------------------------------------------------------------------
// CUDA kernel.

template <class T, int A>
__global__ void bias_act_kernel(bias_act_kernel_params p)
{
    typedef typename InternalType<T>::scalar_t scalar_t;
    int G                 = p.grad;
    scalar_t alpha        = (scalar_t)p.alpha;
    scalar_t gain         = (scalar_t)p.gain;
    scalar_t clamp        = (scalar_t)p.clamp;
    scalar_t one          = (scalar_t)1;
    scalar_t two          = (scalar_t)2;
    scalar_t expRange     = (scalar_t)80;
    scalar_t halfExpRange = (scalar_t)40;
    scalar_t seluScale    = (scalar_t)1.0507009873554804934193349852946;
    scalar_t seluAlpha    = (scalar_t)1.6732632423543772848170429916717;

    // Loop over elements.
    int xi = blockIdx.x * p.loopX * blockDim.x + threadIdx.x;
    for (int loopIdx = 0; loopIdx < p.loopX && xi < p.sizeX; loopIdx++, xi += blockDim.x)
    {
        // Load.
        scalar_t x = (scalar_t)((const T*)p.x)[xi];
        scalar_t b = (p.b) ? (scalar_t)((const T*)p.b)[(xi / p.stepB) % p.sizeB] : 0;
        scalar_t xref = (p.xref) ? (scalar_t)((const T*)p.xref)[xi] : 0;
        scalar_t yref = (p.yref) ? (scalar_t)((const T*)p.yref)[xi] : 0;
        scalar_t dy = (p.dy) ? (scalar_t)((const T*)p.dy)[xi] : one;
        scalar_t yy = (gain != 0) ? yref / gain : 0;
        scalar_t y = 0;

        // Apply bias.
        ((G == 0) ? x : xref) += b;

        // linear
        if (A == 1)
        {
            if (G == 0) y = x;
            if (G == 1) y = x;
        }

        // relu
        if (A == 2)
        {
            if (G == 0) y = (x > 0) ? x : 0;
            if (G == 1) y = (yy > 0) ? x : 0;
        }

        // lrelu
        if (A == 3)
        {
            if (G == 0) y = (x > 0) ? x : x * alpha;
            if (G == 1) y = (yy > 0) ? x : x * alpha;
        }

        // tanh
        if (A == 4)
        {
            if (G == 0) { scalar_t c = exp(x); scalar_t d = one / c; y = (x < -expRange) ? -one : (x > expRange) ? one : (c - d) / (c + d); }
            if (G == 1) y = x * (one - yy * yy);
            if (G == 2) y = x * (one - yy * yy) * (-two * yy);
        }

        // sigmoid
        if (A == 5)
        {
            if (G == 0) y = (x < -expRange) ? 0 : one / (exp(-x) + one);
            if (G == 1) y = x * yy * (one - yy);
            if (G == 2) y = x * yy * (one - yy) * (one - two * yy);
        }

        // elu
        if (A == 6)
        {
            if (G == 0) y = (x >= 0) ? x : exp(x) - one;
            if (G == 1) y = (yy >= 0) ? x : x * (yy + one);
            if (G == 2) y = (yy >= 0) ? 0 : x * (yy + one);
        }

        // selu
        if (A == 7)
        {
            if (G == 0) y = (x >= 0) ? seluScale * x : (seluScale * seluAlpha) * (exp(x) - one);
            if (G == 1) y = (yy >= 0) ? x * seluScale : x * (yy + seluScale * seluAlpha);
            if (G == 2) y = (yy >= 0) ? 0 : x * (yy + seluScale * seluAlpha);
        }

        // softplus
        if (A == 8)
        {
            if (G == 0) y = (x > expRange) ? x : log(exp(x) + one);
            if (G == 1) y = x * (one - exp(-yy));
            if (G == 2) { scalar_t c = exp(-yy); y = x * c * (one - c); }
        }

        // swish
        if (A == 9)
        {
            if (G == 0)
                y = (x < -expRange) ? 0 : x / (exp(-x) + one);
            else
            {
                scalar_t c = exp(xref);
                scalar_t d = c + one;
                if (G == 1)
                    y = (xref > halfExpRange) ? x : x * c * (xref + d) / (d * d);
                else
                    y = (xref > halfExpRange) ? 0 : x * c * (xref * (two - d) + two * d) / (d * d * d);
                yref = (xref < -expRange) ? 0 : xref / (exp(-xref) + one) * gain;
            }
        }

        // Apply gain.
        y *= gain * dy;

        // Clamp.
        if (clamp >= 0)
        {
            if (G == 0)
                y = (y > -clamp & y < clamp) ? y : (y >= 0) ? clamp : -clamp;
            else
                y = (yref > -clamp & yref < clamp) ? y : 0;
        }

        // Store.
        ((T*)p.y)[xi] = (T)y;
    }
}

//------------------------------------------------------------------------
// CUDA kernel selection.

template <class T> void* choose_bias_act_kernel(const bias_act_kernel_params& p)
{
    if (p.act == 1) return (void*)bias_act_kernel<T, 1>;
    if (p.act == 2) return (void*)bias_act_kernel<T, 2>;
    if (p.act == 3) return (void*)bias_act_kernel<T, 3>;
    if (p.act == 4) return (void*)bias_act_kernel<T, 4>;
    if (p.act == 5) return (void*)bias_act_kernel<T, 5>;
    if (p.act == 6) return (void*)bias_act_kernel<T, 6>;
    if (p.act == 7) return (void*)bias_act_kernel<T, 7>;
    if (p.act == 8) return (void*)bias_act_kernel<T, 8>;
    if (p.act == 9) return (void*)bias_act_kernel<T, 9>;
    return NULL;
}

//------------------------------------------------------------------------
// Template specializations.

template void* choose_bias_act_kernel<double>       (const bias_act_kernel_params& p);
template void* choose_bias_act_kernel<float>        (const bias_act_kernel_params& p);
template void* choose_bias_act_kernel<c10::Half>    (const bias_act_kernel_params& p);

//------------------------------------------------------------------------
